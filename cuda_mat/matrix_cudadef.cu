#include <iostream>
#include <cstdlib>
#include <chrono>
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstring>
#include <cmath>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file, const int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        exit(1);
    }
}

struct Matrix_CU {
    int row;
    int col;
    float* data;

    Matrix_CU(int r, int c) : row(r), col(c) {
        data = new float[r * c]();
    }
    // 删除拷贝构造函数和拷贝赋值运算符
    Matrix_CU(const Matrix_CU& other) : row(other.row), col(other.col) {
        data = new float[row * col];
        std::memcpy(data, other.data, row * col * sizeof(float));
    }

    // 拷贝赋值运算符
    Matrix_CU& operator=(const Matrix_CU& other) {
        if (this != &other) {
            delete[] data;
            row = other.row;
            col = other.col;
            data = new float[row * col];
            std::memcpy(data, other.data, row * col * sizeof(float));
        }
        return *this;
    }
    ~Matrix_CU() {
        delete[] data;
    }

    void randomInit() {
        for (int i = 0; i < row * col; ++i) {
            data[i] = static_cast<float>(rand()) / RAND_MAX;
        }
    }

    void printFirstElement() const {
        std::cout << "First element: " << data[0] << std::endl;
    }
    void printSubMatrix(int numRows, int numCols) const {
    for (int i = 0; i < numRows; ++i) {
        for (int j = 0; j < numCols; ++j) {
            std::cout << data[i * col + j] << " ";
        }
        std::cout << std::endl;
        }
    }
    
};
float getMaxValue(const Matrix_CU& matrix) {
    float maxVal = matrix.data[0];
    for (int i = 1; i < matrix.row * matrix.col; ++i) {
            if (matrix.data[i] > maxVal) {
                maxVal = matrix.data[i];
            }
        }
        return maxVal;
    }
float getMinValue(const Matrix_CU& matrix) {
    float minVal = matrix.data[0];
    for (int i = 1; i < matrix.row * matrix.col; ++i) {
        if (matrix.data[i] < minVal) {
            minVal = matrix.data[i];
        }
    }
    return minVal;
}
float getAverageValue(const Matrix_CU& matrix) {
    float sum = 0.0f;
    for (int i = 0; i < matrix.row * matrix.col; ++i) {
        sum += matrix.data[i];
    }
    return sum / (matrix.row * matrix.col);
}
float getSum(const Matrix_CU& matrix) {
    float sum = 0.0f;
    for (int i = 0; i < matrix.row * matrix.col; ++i) {
        sum += matrix.data[i];
    }
    return sum;
}

float getStandardDeviation(const Matrix_CU& matrix) {
    float mean = getAverageValue(matrix);
    float sumSquaredDifferences = 0.0f;
    
    for (int i = 0; i < matrix.row * matrix.col; ++i) {
        float diff = matrix.data[i] - mean;
        sumSquaredDifferences += diff * diff;
    }

    return sqrt(sumSquaredDifferences / (matrix.row * matrix.col));
}

__global__ void matrixAddKernel(const float* A, const float* B, float* C, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx < rows && idy < cols) {
        int linear_idx = idx * cols + idy;
        C[linear_idx] = A[linear_idx] + B[linear_idx];
    }
}

// 修改为直接操作输出矩阵，避免拷贝
void matrixAddCUDA(const Matrix_CU& A, const Matrix_CU& B, Matrix_CU& C) {
    assert(A.row == B.row && A.col == B.col);
    assert(A.row == C.row && A.col == C.col);
    
    const int rows = A.row;
    const int cols = A.col;
    size_t size = rows * cols * sizeof(float);
    
    float *d_A, *d_B, *d_C;
    
    // 1. 分配设备内存
    CHECK_CUDA_ERROR(hipMalloc(&d_A, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_B, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_C, size));

    // 2. 拷贝数据到设备
    CHECK_CUDA_ERROR(hipMemcpy(d_A, A.data, size, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_B, B.data, size, hipMemcpyHostToDevice));

    // 3. 配置内核参数
    dim3 blockSize(16, 16); // 256 threads per block
    dim3 gridSize((cols + blockSize.x - 1) / blockSize.x,
                 (rows + blockSize.y - 1) / blockSize.y);
    auto start = std::chrono::high_resolution_clock::now();
    // 4. 启动核函数
    matrixAddKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, rows, cols);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "GPU执行时间: " << elapsed.count() * 1000 << " ms\n";
    CHECK_CUDA_ERROR(hipGetLastError()); // 检查内核启动错误
    
    // 5. 拷贝结果回主机
    CHECK_CUDA_ERROR(hipMemcpy(C.data, d_C, size, hipMemcpyDeviceToHost));

    // 6. 释放设备内存
    CHECK_CUDA_ERROR(hipFree(d_A));
    CHECK_CUDA_ERROR(hipFree(d_B));
    CHECK_CUDA_ERROR(hipFree(d_C));
}

Matrix_CU matrixAddCPU(const Matrix_CU& A, const Matrix_CU& B) {
    assert(A.row == B.row && A.col == B.col);
    
    Matrix_CU C(A.row, A.col);
    
    auto start = std::chrono::high_resolution_clock::now();
    
    for (int i = 0; i < A.row * A.col; ++i) {
        C.data[i] = A.data[i] + B.data[i];
    }
    
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "CPU执行时间: " << elapsed.count() * 1000 << " ms\n";
    
    return C;
}

void testMatrix_CUAddition() {
    const int rows = 10240;
    const int cols = 10240;
    std::cout << "测试矩阵大小: " << rows << "x" << cols << "\n";
    
    Matrix_CU A(rows, cols);
    Matrix_CU B(rows, cols);
    Matrix_CU C_gpu(rows, cols);  // GPU结果矩阵
    
    A.randomInit();
    B.randomInit();
    
    std::cout << "矩阵A: ";
    A.printFirstElement();
    std::cout << "矩阵B: ";
    B.printFirstElement();

    std::cout << "\nCPU版本:\n";
    Matrix_CU C_cpu = matrixAddCPU(A, B);
    std::cout << "CPU结果: ";
    C_cpu.printFirstElement();

    std::cout << "\nGPU版本:\n";
    matrixAddCUDA(A, B, C_gpu);
    std::cout << "GPU结果: ";
    C_gpu.printFirstElement();
    std::cout<< "标准差:" << getStandardDeviation(C_gpu);
    std::cout << "最大值: " << getMaxValue(C_gpu) << std::endl;
    std::cout << "最小值: " << getMinValue(C_gpu) << std::endl;
    std::cout << "平均值: " << getAverageValue(C_gpu) << std::endl;
    std::cout << "和: " << getSum(C_gpu) << std::endl;
    std::cout << "标准差: " << getStandardDeviation(C_gpu) << std::endl;
    // 验证结果
    bool correct = true;
    for (int i = 0; i < 10 && correct; ++i) {
        if (fabs(C_cpu.data[i] - C_gpu.data[i]) > 1e-5) {
            std::cout << "结果不匹配在索引 " << i 
                      << ": CPU=" << C_cpu.data[i] 
                      << ", GPU=" << C_gpu.data[i] << "\n";
            correct = false;
        }
    }
    
    if (correct) {
        std::cout << "前10个元素验证成功!\n";
    }
}

int main() {
    // 初始化CUDA设备
    hipDeviceProp_t prop;
    CHECK_CUDA_ERROR(hipGetDeviceProperties(&prop, 0));
    std::cout << "使用GPU: " << prop.name << "\n";
    
    testMatrix_CUAddition();
    return 0;
}