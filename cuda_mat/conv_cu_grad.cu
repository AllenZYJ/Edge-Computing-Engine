#include "hip/hip_runtime.h"
// Test function for convolution
#include <iostream>
#include <cstdlib>
#include <chrono>
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstring>
#include <cmath>
#include <functional>
#include "mat_grad.cu"
void testConvolution() {
    std::cout << "\n====== Convolution Test ======\n";
    
    // Configuration
    int batch_size = 2;
    int in_channels = 3;
    int in_height = 5;
    int in_width = 5;
    int out_channels = 2;
    int kernel_size = 3;
    int stride = 1;
    int padding = 1;
    
    // Expected output dimensions
    int out_height = ((in_height + 2 * padding - kernel_size) / stride) + 1;  // Should be 5
    int out_width = ((in_width + 2 * padding - kernel_size) / stride) + 1;    // Should be 5
    
    std::cout << "Configuration:\n";
    std::cout << "  Input: [" << batch_size << ", " << in_channels << ", " << in_height << ", " << in_width << "]\n";
    std::cout << "  Kernel: [" << out_channels << ", " << in_channels << ", " << kernel_size << ", " << kernel_size << "]\n";
    std::cout << "  Stride: [" << stride << ", " << stride << "], Padding: [" << padding << ", " << padding << "]\n";
    std::cout << "  Expected Output: [" << batch_size << ", " << out_channels << ", " << out_height << ", " << out_width << "]\n";
    
    ComputeGraph graph;
    
    // Create input tensor: [batch_size * in_channels, in_height * in_width]
    ComputeNode* input = graph.addInput(batch_size * in_channels, in_height * in_width);
    input->batchSize = batch_size;
    
    // Create kernel tensor: [out_channels, in_channels * kernel_size * kernel_size]
    ComputeNode* kernel = graph.addInput(out_channels, in_channels * kernel_size * kernel_size);
    
    // Initialize input with a pattern that increases along width, then height, then channel
    for (int b = 0; b < batch_size; b++) {
        for (int c = 0; c < in_channels; c++) {
            for (int h = 0; h < in_height; h++) {
                for (int w = 0; w < in_width; w++) {
                    int flat_idx = (b * in_channels + c) * (in_height * in_width) + (h * in_width + w);
                    // Simple pattern: value = (c+1) * 0.1 + h * 0.01 + w * 0.001
                    input->value.data[flat_idx] = (c + 1) * 0.1f + h * 0.01f + w * 0.001f;
                }
            }
        }
    }
    
    // Initialize kernel with identity-like pattern for testing
    for (int oc = 0; oc < out_channels; oc++) {
        for (int ic = 0; ic < in_channels; ic++) {
            for (int kh = 0; kh < kernel_size; kh++) {
                for (int kw = 0; kw < kernel_size; kw++) {
                    int flat_idx = oc * (in_channels * kernel_size * kernel_size) + 
                                 (ic * kernel_size * kernel_size + kh * kernel_size + kw);
                    
                    // Identity-like kernel for the center position, zeros elsewhere
                    // For convolution, this should preserve the input features with some edge effects
                    if (kh == kernel_size/2 && kw == kernel_size/2 && ic == oc % in_channels) {
                        kernel->value.data[flat_idx] = 1.0f;
                    } else {
                        kernel->value.data[flat_idx] = 0.0f;
                    }
                }
            }
        }
    }
    
    // Create convolution node
    ComputeNode* conv = graph.addConv2D(input, kernel, in_height, in_width, in_channels, 
                                      stride, stride, padding, padding);
    
    // Forward pass
    auto start = std::chrono::high_resolution_clock::now();
    graph.forward();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> forward_duration = end - start;
    
    std::cout << "\nForward Pass Results:\n";
    std::cout << "  Convolution output shape: [" << conv->value.row << ", " << conv->value.col << "]\n";
    std::cout << "  Time: " << forward_duration.count() << " ms\n";
    
    // Print sample of the output (first batch, first channel)
    std::cout << "\nOutput sample (first batch, first channel):\n";
    for (int h = 0; h < out_height; h++) {
        std::cout << "  ";
        for (int w = 0; w < out_width; w++) {
            int flat_idx = 0 * (out_height * out_width) + (h * out_width + w);
            printf("%.4f ", conv->value.data[flat_idx]);
        }
        std::cout << std::endl;
    }
    
    // Backward pass
    start = std::chrono::high_resolution_clock::now();
    graph.backward(conv);
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> backward_duration = end - start;
    
    std::cout << "\nBackward Pass Results:\n";
    std::cout << "  Time: " << backward_duration.count() << " ms\n";
    
    // Print sample of input gradients (first batch, first channel)
    std::cout << "\nInput gradient sample (first batch, first channel):\n";
    for (int h = 0; h < 3; h++) {
        std::cout << "  ";
        for (int w = 0; w < 3; w++) {
            int flat_idx = 0 * (in_height * in_width) + (h * in_width + w);
            printf("%.4f ", input->grad.data[flat_idx]);
        }
        std::cout << std::endl;
    }
    
    // Print sample of kernel gradients (first output channel, first input channel)
    std::cout << "\nKernel gradient sample (first output channel, first input channel):\n";
    for (int kh = 0; kh < kernel_size; kh++) {
        std::cout << "  ";
        for (int kw = 0; kw < kernel_size; kw++) {
            int flat_idx = 0 * (in_channels * kernel_size * kernel_size) + 
                         (0 * kernel_size * kernel_size + kh * kernel_size + kw);
            printf("%.4f ", kernel->grad.data[flat_idx]);
        }
        std::cout << std::endl;
    }
    
    std::cout << "\nConvolution Test Completed\n";
}

// CPU reference implementation for verification
void conv2dCPU(const float* input, const float* kernel, float* output,
              int batch_size, int in_channels, int out_channels,
              int in_height, int in_width, int kernel_height, int kernel_width,
              int out_height, int out_width, int stride_height, int stride_width,
              int pad_height, int pad_width) {
    
    for (int n = 0; n < batch_size; n++) {
        for (int c = 0; c < out_channels; c++) {
            for (int h = 0; h < out_height; h++) {
                for (int w = 0; w < out_width; w++) {
                    float sum = 0.0f;
                    
                    for (int ic = 0; ic < in_channels; ic++) {
                        for (int kh = 0; kh < kernel_height; kh++) {
                            for (int kw = 0; kw < kernel_width; kw++) {
                                int ih = h * stride_height + kh - pad_height;
                                int iw = w * stride_width + kw - pad_width;
                                
                                if (ih >= 0 && ih < in_height && iw >= 0 && iw < in_width) {
                                    int input_idx = ((n * in_channels + ic) * in_height + ih) * in_width + iw;
                                    int kernel_idx = ((c * in_channels + ic) * kernel_height + kh) * kernel_width + kw;
                                    
                                    sum += input[input_idx] * kernel[kernel_idx];
                                }
                            }
                        }
                    }
                    
                    int output_idx = ((n * out_channels + c) * out_height + h) * out_width + w;
                    output[output_idx] = sum;
                }
            }
        }
    }
}

// Test function with CPU validation
void testConvolutionWithValidation() {
    std::cout << "\n====== Convolution Test with CPU Validation ======\n";
    
    // Configuration (smaller for CPU comparison)
    int batch_size = 2;
    int in_channels = 2;
    int in_height = 4;
    int in_width = 4;
    int out_channels = 2;
    int kernel_size = 3;
    int stride = 1;
    int padding = 1;
    
    // Expected output dimensions
    int out_height = ((in_height + 2 * padding - kernel_size) / stride) + 1;
    int out_width = ((in_width + 2 * padding - kernel_size) / stride) + 1;
    
    ComputeGraph graph;
    
    // Create input tensor
    ComputeNode* input = graph.addInput(batch_size * in_channels, in_height * in_width);
    input->batchSize = batch_size;
    
    // Create kernel tensor
    ComputeNode* kernel = graph.addInput(out_channels, in_channels * kernel_size * kernel_size);
    
    // Initialize with random values for better test
    for (int i = 0; i < input->value.row * input->value.col; i++) {
        input->value.data[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    
    for (int i = 0; i < kernel->value.row * kernel->value.col; i++) {
        kernel->value.data[i] = static_cast<float>(rand()) / RAND_MAX * 0.1f;
    }
    
    // Create convolution node
    ComputeNode* conv = graph.addConv2D(input, kernel, in_height, in_width, in_channels, 
                                      stride, stride, padding, padding);
    
    // Forward pass
    graph.forward();
    
    // CPU reference calculation
    std::vector<float> input_reshaped(batch_size * in_channels * in_height * in_width);
    std::vector<float> kernel_reshaped(out_channels * in_channels * kernel_size * kernel_size);
    std::vector<float> output_cpu(batch_size * out_channels * out_height * out_width, 0.0f);
    
    // Reshape input for CPU calculation
    for (int n = 0; n < batch_size; n++) {
        for (int c = 0; c < in_channels; c++) {
            for (int h = 0; h < in_height; h++) {
                for (int w = 0; w < in_width; w++) {
                    int flat_idx = (n * in_channels + c) * (in_height * in_width) + (h * in_width + w);
                    int tensor_idx = ((n * in_channels + c) * in_height + h) * in_width + w;
                    input_reshaped[tensor_idx] = input->value.data[flat_idx];
                }
            }
        }
    }
    
    // Reshape kernel for CPU calculation
    for (int oc = 0; oc < out_channels; oc++) {
        for (int ic = 0; ic < in_channels; ic++) {
            for (int kh = 0; kh < kernel_size; kh++) {
                for (int kw = 0; kw < kernel_size; kw++) {
                    int flat_idx = oc * (in_channels * kernel_size * kernel_size) + 
                                 (ic * kernel_size * kernel_size + kh * kernel_size + kw);
                    int tensor_idx = ((oc * in_channels + ic) * kernel_size + kh) * kernel_size + kw;
                    kernel_reshaped[tensor_idx] = kernel->value.data[flat_idx];
                }
            }
        }
    }
    
    // CPU convolution
    conv2dCPU(input_reshaped.data(), kernel_reshaped.data(), output_cpu.data(),
             batch_size, in_channels, out_channels,
             in_height, in_width, kernel_size, kernel_size,
             out_height, out_width, stride, stride,
             padding, padding);
    
    // Compare results
    std::vector<float> output_gpu(batch_size * out_channels * out_height * out_width);
    for (int n = 0; n < batch_size; n++) {
        for (int c = 0; c < out_channels; c++) {
            for (int h = 0; h < out_height; h++) {
                for (int w = 0; w < out_width; w++) {
                    int flat_idx = (n * out_channels + c) * (out_height * out_width) + (h * out_width + w);
                    int tensor_idx = ((n * out_channels + c) * out_height + h) * out_width + w;
                    output_gpu[tensor_idx] = conv->value.data[flat_idx];
                }
            }
        }
    }
    
    // Calculate error
    float max_diff = 0.0f;
    float avg_diff = 0.0f;
    
    for (int i = 0; i < output_cpu.size(); i++) {
        float diff = std::abs(output_cpu[i] - output_gpu[i]);
        max_diff = std::max(max_diff, diff);
        avg_diff += diff;
    }
    avg_diff /= output_cpu.size();
    
    std::cout << "Validation Results:\n";
    std::cout << "  Maximum absolute difference: " << max_diff << std::endl;
    std::cout << "  Average absolute difference: " << avg_diff << std::endl;
    std::cout << "  Validation " << (max_diff < 1e-4 ? "PASSED" : "FAILED") << std::endl;
    
    std::cout << "\nConvolution Validation Completed\n";
}

// Update main function to test convolution
int main() {
    // Test convolution
    testConvolution();
    
    // Test with CPU validation
    testConvolutionWithValidation();
    
    // Original code can follow here
    // ...
    
    return 0;
}