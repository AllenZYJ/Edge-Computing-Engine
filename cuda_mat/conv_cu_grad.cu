#include <iostream>
#include <cstdlib>
#include <chrono>
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstring>
#include <cmath>
#include <functional>
#include <vector>
#include <iomanip>
#include "mat_grad.cu"

// 已有的CPU卷积实现
void conv2dCPU(const float* input, const float* kernel, float* output,
              int batch_size, int in_channels, int out_channels,
              int in_height, int in_width, int kernel_height, int kernel_width,
              int out_height, int out_width, int stride_height, int stride_width,
              int pad_height, int pad_width) {
    
    for (int n = 0; n < batch_size; n++) {
        for (int c = 0; c < out_channels; c++) {
            for (int h = 0; h < out_height; h++) {
                for (int w = 0; w < out_width; w++) {
                    float sum = 0.0f;
                    
                    for (int ic = 0; ic < in_channels; ic++) {
                        for (int kh = 0; kh < kernel_height; kh++) {
                            for (int kw = 0; kw < kernel_width; kw++) {
                                int ih = h * stride_height + kh - pad_height;
                                int iw = w * stride_width + kw - pad_width;
                                
                                if (ih >= 0 && ih < in_height && iw >= 0 && iw < in_width) {
                                    int input_idx = ((n * in_channels + ic) * in_height + ih) * in_width + iw;
                                    int kernel_idx = ((c * in_channels + ic) * kernel_height + kh) * kernel_width + kw;
                                    
                                    sum += input[input_idx] * kernel[kernel_idx];
                                }
                            }
                        }
                    }
                    
                    int output_idx = ((n * out_channels + c) * out_height + h) * out_width + w;
                    output[output_idx] = sum;
                }
            }
        }
    }
}

// 新增的速度比较函数
void compareConvolutionSpeed() {
    std::cout << "\n====== CPU vs GPU Convolution Speed Comparison ======\n";
    
    // 定义不同的测试配置
    struct TestConfig {
        int batch_size;
        int in_channels;
        int in_height;
        int in_width;
        int out_channels;
        int kernel_size;
        int stride;
        int padding;
    };
    
    std::vector<TestConfig> configs = {
        {1, 3, 32, 32, 16, 3, 1, 1},      // 小尺寸
        {4, 3, 64, 64, 32, 3, 1, 1},      // 中等尺寸
        {16, 3, 128, 128, 64, 3, 1, 1},   // 大尺寸
        {32, 3, 224, 224, 64, 3, 1, 1},   // 类似VGG的输入
        {32, 64, 56, 56, 128, 3, 1, 1},   // 类似中层特征图
        {16, 128, 28, 28, 256, 3, 1, 1},  // 类似深层特征图
    };
    
    // 表头
    std::cout << std::left << std::setw(20) << "配置" 
              << std::setw(15) << "输入尺寸" 
              << std::setw(15) << "内核尺寸" 
              << std::setw(15) << "CPU时间(ms)" 
              << std::setw(15) << "GPU时间(ms)" 
              << std::setw(15) << "加速比(x)" 
              << std::endl;
    std::cout << std::string(95, '-') << std::endl;
    
    // 对每个配置进行测试
    for (size_t i = 0; i < configs.size(); i++) {
        const auto& cfg = configs[i];
        
        // 计算输出尺寸
        int out_height = ((cfg.in_height + 2 * cfg.padding - cfg.kernel_size) / cfg.stride) + 1;
        int out_width = ((cfg.in_width + 2 * cfg.padding - cfg.kernel_size) / cfg.stride) + 1;
        
        // 创建计算图和节点(GPU)
        ComputeGraph graph;
        ComputeNode* input = graph.addInput(cfg.batch_size * cfg.in_channels, cfg.in_height * cfg.in_width);
        input->batchSize = cfg.batch_size;
        ComputeNode* kernel = graph.addInput(cfg.out_channels, cfg.in_channels * cfg.kernel_size * cfg.kernel_size);
        
        // 随机初始化值
        for (int i = 0; i < input->value.row * input->value.col; i++) {
            input->value.data[i] = static_cast<float>(rand()) / RAND_MAX;
        }
        
        for (int i = 0; i < kernel->value.row * kernel->value.col; i++) {
            kernel->value.data[i] = static_cast<float>(rand()) / RAND_MAX * 0.1f;
        }
        
        // 创建卷积节点
        ComputeNode* conv = graph.addConv2D(input, kernel, cfg.in_height, cfg.in_width, cfg.in_channels, 
                                          cfg.stride, cfg.stride, cfg.padding, cfg.padding);
        
        // 为CPU实现准备数据
        std::vector<float> input_cpu(cfg.batch_size * cfg.in_channels * cfg.in_height * cfg.in_width);
        std::vector<float> kernel_cpu(cfg.out_channels * cfg.in_channels * cfg.kernel_size * cfg.kernel_size);
        std::vector<float> output_cpu(cfg.batch_size * cfg.out_channels * out_height * out_width, 0.0f);
        
        // 重塑输入数据为CPU格式
        for (int n = 0; n < cfg.batch_size; n++) {
            for (int c = 0; c < cfg.in_channels; c++) {
                for (int h = 0; h < cfg.in_height; h++) {
                    for (int w = 0; w < cfg.in_width; w++) {
                        int flat_idx = (n * cfg.in_channels + c) * (cfg.in_height * cfg.in_width) + (h * cfg.in_width + w);
                        int tensor_idx = ((n * cfg.in_channels + c) * cfg.in_height + h) * cfg.in_width + w;
                        input_cpu[tensor_idx] = input->value.data[flat_idx];
                    }
                }
            }
        }
        
        // 重塑内核数据为CPU格式
        for (int oc = 0; oc < cfg.out_channels; oc++) {
            for (int ic = 0; ic < cfg.in_channels; ic++) {
                for (int kh = 0; kh < cfg.kernel_size; kh++) {
                    for (int kw = 0; kw < cfg.kernel_size; kw++) {
                        int flat_idx = oc * (cfg.in_channels * cfg.kernel_size * cfg.kernel_size) + 
                                     (ic * cfg.kernel_size * cfg.kernel_size + kh * cfg.kernel_size + kw);
                        int tensor_idx = ((oc * cfg.in_channels + ic) * cfg.kernel_size + kh) * cfg.kernel_size + kw;
                        kernel_cpu[tensor_idx] = kernel->value.data[flat_idx];
                    }
                }
            }
        }
        
        // 计时器
        std::chrono::duration<double, std::milli> gpu_duration;
        std::chrono::duration<double, std::milli> cpu_duration;
        
        // GPU卷积计时（多次运行取平均值）
        const int num_runs = 10;
        auto gpu_start = std::chrono::high_resolution_clock::now();
        for (int run = 0; run < num_runs; run++) {
            graph.forward();
            // 为避免结果被缓存，修改输入小量值
            input->value.data[0] += 1e-5f;
        }
        auto gpu_end = std::chrono::high_resolution_clock::now();
        gpu_duration = (gpu_end - gpu_start) / num_runs;
        
        // CPU卷积计时（对于大尺寸配置减少运行次数）
        int cpu_runs = (i < 3) ? num_runs : 3;  // 大尺寸配置少运行几次
        auto cpu_start = std::chrono::high_resolution_clock::now();
        for (int run = 0; run < cpu_runs; run++) {
            conv2dCPU(input_cpu.data(), kernel_cpu.data(), output_cpu.data(),
                     cfg.batch_size, cfg.in_channels, cfg.out_channels,
                     cfg.in_height, cfg.in_width, cfg.kernel_size, cfg.kernel_size,
                     out_height, out_width, cfg.stride, cfg.stride,
                     cfg.padding, cfg.padding);
        }
        auto cpu_end = std::chrono::high_resolution_clock::now();
        cpu_duration = (cpu_end - cpu_start) / cpu_runs;
        
        // 计算加速比
        double speedup = cpu_duration.count() / gpu_duration.count();
        
        // 打印结果
        std::cout << std::left << std::setw(20) << "配置 " + std::to_string(i+1)
                  << std::setw(15) << cfg.batch_size * cfg.in_channels * cfg.in_height * cfg.in_width
                  << std::setw(15) << cfg.out_channels * cfg.in_channels * cfg.kernel_size * cfg.kernel_size
                  << std::setw(15) << std::fixed << std::setprecision(2) << cpu_duration.count()
                  << std::setw(15) << std::fixed << std::setprecision(2) << gpu_duration.count()
                  << std::setw(15) << std::fixed << std::setprecision(2) << speedup
                  << std::endl;
    }
    
    std::cout << "\nCPU和GPU卷积速度比较完成\n";
}

// 更新main函数以包含速度比较
int main() {
    // 设置随机数种子
    srand(42);
    
    // 运行速度比较测试
// 配置              输入尺寸   内核尺寸   CPU时间(ms)  GPU时间(ms)  加速比(x)
// -----------------------------------------------------------------------------------------------
// 配置 1            3072           432            2.01           5.70           0.35
// 配置 2            49152          864            65.87          6.36           10.35
// 配置 3            786432         1728           2126.64        212.35         10.01
// 配置 4            4816896        1728           12969.35       1319.69        9.83
// 配置 5            6422528        73728          33261.28       387.02         85.94
// 配置 6            1605632        294912         16521.94       140.25         117.80
    compareConvolutionSpeed();
    
    return 0;
}