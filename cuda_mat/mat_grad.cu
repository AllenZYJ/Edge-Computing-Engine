#include "hip/hip_runtime.h"
#include "matrix_cudadef.cu"
#include <iostream>
#include <cstdlib>
#include <chrono>
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstring>
#include <cmath>
#include <functional>
// 自动微分节点类型枚举
enum OpType {
    OP_INPUT,    // 输入节点
    OP_ADD,      // 加法
    OP_SUB,      // 减法
    OP_MUL,      // 元素级乘法
    OP_DIV,      // 元素级除法
    OP_MATMUL,   // 矩阵乘法
    OP_RELU,     // ReLU激活函数
    OP_SIGMOID,  // Sigmoid激活函数
    OP_TANH,     // Tanh激活函数
    OP_SOFTMAX   // Softmax函数
};

// 计算图节点
struct ComputeNode {
    OpType op;                     // 操作类型
    Matrix_CU value;               // 前向传播值
    Matrix_CU grad;                // 梯度值
    std::vector<ComputeNode*> inputs;  // 输入节点
    int batchSize;                 // 批处理大小
    
    ComputeNode(int rows, int cols, int batch = 1) : 
        op(OP_INPUT), 
        value(rows * batch, cols), 
        grad(rows * batch, cols), 
        batchSize(batch) {}
    
    // 防止内存泄漏
    ~ComputeNode() {
        // 析构函数会自动处理value和grad
    }
    
    // 重置梯度
    void zeroGrad() {
        std::memset(grad.data, 0, grad.row * grad.col * sizeof(float));
    }
};

// CUDA核函数：ReLU前向传播
__global__ void reluForwardKernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = fmaxf(0.0f, input[idx]);
    }
}

// CUDA核函数：ReLU反向传播
__global__ void reluBackwardKernel(const float* outGrad, const float* input, float* inGrad, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        inGrad[idx] = (input[idx] > 0.0f) ? outGrad[idx] : 0.0f;
    }
}

// CUDA核函数：Sigmoid前向传播
__global__ void sigmoidForwardKernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = 1.0f / (1.0f + expf(-input[idx]));
    }
}

// CUDA核函数：Sigmoid反向传播
__global__ void sigmoidBackwardKernel(const float* outGrad, const float* output, float* inGrad, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // sigmoid的导数是 sigmoid(x) * (1 - sigmoid(x))
        inGrad[idx] = outGrad[idx] * output[idx] * (1.0f - output[idx]);
    }
}

// CUDA核函数：矩阵乘法前向传播 (C = A * B)
__global__ void matmulForwardKernel(const float* A, const float* B, float* C, 
                                   int A_rows, int A_cols, int B_cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < A_rows && col < B_cols) {
        float sum = 0.0f;
        for (int k = 0; k < A_cols; ++k) {
            sum += A[row * A_cols + k] * B[k * B_cols + col];
        }
        C[row * B_cols + col] = sum;
    }
}

// CUDA核函数：矩阵乘法反向传播 - A的梯度
__global__ void matmulBackwardAKernel(const float* outGrad, const float* B, float* inGradA,
                                     int outGrad_rows, int B_rows, int B_cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < outGrad_rows && col < B_rows) {
        float sum = 0.0f;
        for (int k = 0; k < B_cols; ++k) {
            sum += outGrad[row * B_cols + k] * B[col * B_cols + k];
        }
        inGradA[row * B_rows + col] = sum;
    }
}

// CUDA核函数：矩阵乘法反向传播 - B的梯度
__global__ void matmulBackwardBKernel(const float* A, const float* outGrad, float* inGradB,
                                     int A_rows, int A_cols, int outGrad_cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < A_cols && col < outGrad_cols) {
        float sum = 0.0f;
        for (int k = 0; k < A_rows; ++k) {
            sum += A[k * A_cols + row] * outGrad[k * outGrad_cols + col];
        }
        inGradB[row * outGrad_cols + col] = sum;
    }
}

// 为加法和减法操作修改反向传播实现
void addBackwardCUDA(const Matrix_CU& outGrad, Matrix_CU& inGradA, Matrix_CU& inGradB) {
    assert(outGrad.row == inGradA.row && outGrad.col == inGradA.col);
    assert(outGrad.row == inGradB.row && outGrad.col == inGradB.col);
    
    // 加法的梯度直接复制到两个输入（主机内存到主机内存）
    const int size = outGrad.row * outGrad.col * sizeof(float);
    std::memcpy(inGradA.data, outGrad.data, size);
    std::memcpy(inGradB.data, outGrad.data, size);
}
// 添加一个全局的CUDA核函数，替代lambda函数
__global__ void negGradKernel(const float* outGrad, float* inGrad, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        inGrad[idx] = -outGrad[idx];
    }
}
void subBackwardCUDA(const Matrix_CU& outGrad, Matrix_CU& inGradA, Matrix_CU& inGradB) {
    assert(outGrad.row == inGradA.row && outGrad.col == inGradA.col);
    assert(outGrad.row == inGradB.row && outGrad.col == inGradB.col);
    
    const int rows = outGrad.row;
    const int cols = outGrad.col;
    const int totalElements = rows * cols;
    const size_t size = totalElements * sizeof(float);
    
    // 对于A: 直接复制梯度（主机内存到主机内存）
    std::memcpy(inGradA.data, outGrad.data, size);
    
    // 对于B: 计算负梯度
    float *d_outGrad, *d_inGradB;
    CHECK_CUDA_ERROR(hipMalloc(&d_outGrad, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_inGradB, size));
    
    CHECK_CUDA_ERROR(hipMemcpy(d_outGrad, outGrad.data, size, hipMemcpyHostToDevice));
    
    int blockSize = 256;
    int numBlocks = (totalElements + blockSize - 1) / blockSize;
    
    negGradKernel<<<numBlocks, blockSize>>>(d_outGrad, d_inGradB, totalElements);
    
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipMemcpy(inGradB.data, d_inGradB, size, hipMemcpyDeviceToHost));
    
    CHECK_CUDA_ERROR(hipFree(d_outGrad));
    CHECK_CUDA_ERROR(hipFree(d_inGradB));
}




// 元素级乘法的反向传播CUDA实现
__global__ void mulBackwardKernel(const float* outGrad, const float* A, const float* B, 
                                 float* inGradA, float* inGradB, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // 对于A: ∂L/∂A = ∂L/∂(A*B) * ∂(A*B)/∂A = ∂L/∂(A*B) * B
        inGradA[idx] = outGrad[idx] * B[idx];
        
        // 对于B: ∂L/∂B = ∂L/∂(A*B) * ∂(A*B)/∂B = ∂L/∂(A*B) * A
        inGradB[idx] = outGrad[idx] * A[idx];
    }
}

void mulBackwardCUDA(const Matrix_CU& outGrad, const Matrix_CU& A, const Matrix_CU& B, 
                    Matrix_CU& inGradA, Matrix_CU& inGradB) {
    assert(outGrad.row == A.row && outGrad.col == A.col);
    assert(A.row == B.row && A.col == B.col);
    assert(outGrad.row == inGradA.row && outGrad.col == inGradA.col);
    assert(outGrad.row == inGradB.row && outGrad.col == inGradB.col);
    
    const int rows = outGrad.row;
    const int cols = outGrad.col;
    const int totalElements = rows * cols;
    const size_t size = totalElements * sizeof(float);
    
    float *d_outGrad, *d_A, *d_B, *d_inGradA, *d_inGradB;
    
    CHECK_CUDA_ERROR(hipMalloc(&d_outGrad, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_A, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_B, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_inGradA, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_inGradB, size));
    
    CHECK_CUDA_ERROR(hipMemcpy(d_outGrad, outGrad.data, size, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_A, A.data, size, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_B, B.data, size, hipMemcpyHostToDevice));
    
    int blockSize = 256;
    int numBlocks = (totalElements + blockSize - 1) / blockSize;
    
    mulBackwardKernel<<<numBlocks, blockSize>>>(d_outGrad, d_A, d_B, d_inGradA, d_inGradB, totalElements);
    
    CHECK_CUDA_ERROR(hipGetLastError());
    
    CHECK_CUDA_ERROR(hipMemcpy(inGradA.data, d_inGradA, size, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(inGradB.data, d_inGradB, size, hipMemcpyDeviceToHost));
    
    CHECK_CUDA_ERROR(hipFree(d_outGrad));
    CHECK_CUDA_ERROR(hipFree(d_A));
    CHECK_CUDA_ERROR(hipFree(d_B));
    CHECK_CUDA_ERROR(hipFree(d_inGradA));
    CHECK_CUDA_ERROR(hipFree(d_inGradB));
}

// 元素级除法的反向传播CUDA实现
__global__ void divBackwardKernel(const float* outGrad, const float* A, const float* B, 
                                float* inGradA, float* inGradB, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // 对于A: ∂L/∂A = ∂L/∂(A/B) * ∂(A/B)/∂A = ∂L/∂(A/B) * (1/B)
        inGradA[idx] = (B[idx] != 0.0f) ? (outGrad[idx] / B[idx]) : 0.0f;
        
        // 对于B: ∂L/∂B = ∂L/∂(A/B) * ∂(A/B)/∂B = ∂L/∂(A/B) * (-A/B²)
        inGradB[idx] = (B[idx] != 0.0f) ? (outGrad[idx] * (-A[idx] / (B[idx] * B[idx]))) : 0.0f;
    }
}

void divBackwardCUDA(const Matrix_CU& outGrad, const Matrix_CU& A, const Matrix_CU& B, 
                   Matrix_CU& inGradA, Matrix_CU& inGradB) {
    // 与乘法类似，但导数计算不同
    assert(outGrad.row == A.row && outGrad.col == A.col);
    assert(A.row == B.row && A.col == B.col);
    assert(outGrad.row == inGradA.row && outGrad.col == inGradA.col);
    assert(outGrad.row == inGradB.row && outGrad.col == inGradB.col);
    
    const int rows = outGrad.row;
    const int cols = outGrad.col;
    const int totalElements = rows * cols;
    const size_t size = totalElements * sizeof(float);
    
    float *d_outGrad, *d_A, *d_B, *d_inGradA, *d_inGradB;
    
    CHECK_CUDA_ERROR(hipMalloc(&d_outGrad, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_A, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_B, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_inGradA, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_inGradB, size));
    
    CHECK_CUDA_ERROR(hipMemcpy(d_outGrad, outGrad.data, size, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_A, A.data, size, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_B, B.data, size, hipMemcpyHostToDevice));
    
    int blockSize = 256;
    int numBlocks = (totalElements + blockSize - 1) / blockSize;
    
    divBackwardKernel<<<numBlocks, blockSize>>>(d_outGrad, d_A, d_B, d_inGradA, d_inGradB, totalElements);
    
    CHECK_CUDA_ERROR(hipGetLastError());
    
    CHECK_CUDA_ERROR(hipMemcpy(inGradA.data, d_inGradA, size, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(inGradB.data, d_inGradB, size, hipMemcpyDeviceToHost));
    
    CHECK_CUDA_ERROR(hipFree(d_outGrad));
    CHECK_CUDA_ERROR(hipFree(d_A));
    CHECK_CUDA_ERROR(hipFree(d_B));
    CHECK_CUDA_ERROR(hipFree(d_inGradA));
    CHECK_CUDA_ERROR(hipFree(d_inGradB));
}

// 计算图管理类
class ComputeGraph {
public:
    std::vector<ComputeNode*> nodes;
    
    ~ComputeGraph() {
        for (auto node : nodes) {
            delete node;
        }
    }
    
    // 添加输入节点
    ComputeNode* addInput(int rows, int cols, int batchSize = 1) {
        ComputeNode* node = new ComputeNode(rows, cols, batchSize);
        node->op = OP_INPUT;
        nodes.push_back(node);
        return node;
    }
    
    // 添加加法节点
    ComputeNode* addAdd(ComputeNode* a, ComputeNode* b) {
        assert(a->value.row == b->value.row && a->value.col == b->value.col);
        
        ComputeNode* node = new ComputeNode(a->value.row / a->batchSize, a->value.col, a->batchSize);
        node->op = OP_ADD;
        node->inputs.push_back(a);
        node->inputs.push_back(b);
        nodes.push_back(node);
        return node;
    }
    
    // 添加减法节点
    ComputeNode* addSub(ComputeNode* a, ComputeNode* b) {
        assert(a->value.row == b->value.row && a->value.col == b->value.col);
        
        ComputeNode* node = new ComputeNode(a->value.row / a->batchSize, a->value.col, a->batchSize);
        node->op = OP_SUB;
        node->inputs.push_back(a);
        node->inputs.push_back(b);
        nodes.push_back(node);
        return node;
    }
    
    // 添加元素级乘法节点
    ComputeNode* addMul(ComputeNode* a, ComputeNode* b) {
        assert(a->value.row == b->value.row && a->value.col == b->value.col);
        
        ComputeNode* node = new ComputeNode(a->value.row / a->batchSize, a->value.col, a->batchSize);
        node->op = OP_MUL;
        node->inputs.push_back(a);
        node->inputs.push_back(b);
        nodes.push_back(node);
        return node;
    }
    
    // 添加元素级除法节点
    ComputeNode* addDiv(ComputeNode* a, ComputeNode* b) {
        assert(a->value.row == b->value.row && a->value.col == b->value.col);
        
        ComputeNode* node = new ComputeNode(a->value.row / a->batchSize, a->value.col, a->batchSize);
        node->op = OP_DIV;
        node->inputs.push_back(a);
        node->inputs.push_back(b);
        nodes.push_back(node);
        return node;
    }
    
    // 添加矩阵乘法节点
    ComputeNode* addMatMul(ComputeNode* a, ComputeNode* b) {
        assert(a->value.col == b->value.row);
        
        ComputeNode* node = new ComputeNode(a->value.row / a->batchSize, b->value.col, a->batchSize);
        node->op = OP_MATMUL;
        node->inputs.push_back(a);
        node->inputs.push_back(b);
        nodes.push_back(node);
        return node;
    }
    
    // 添加ReLU激活函数节点
    ComputeNode* addReLU(ComputeNode* input) {
        ComputeNode* node = new ComputeNode(input->value.row / input->batchSize, input->value.col, input->batchSize);
        node->op = OP_RELU;
        node->inputs.push_back(input);
        nodes.push_back(node);
        return node;
    }
    
    // 添加Sigmoid激活函数节点
    ComputeNode* addSigmoid(ComputeNode* input) {
        ComputeNode* node = new ComputeNode(input->value.row / input->batchSize, input->value.col, input->batchSize);
        node->op = OP_SIGMOID;
        node->inputs.push_back(input);
        nodes.push_back(node);
        return node;
    }
    
    // 前向传播
    void forward() {
        for (ComputeNode* node : nodes) {
            switch (node->op) {
                case OP_INPUT:
                    // 输入节点不需要计算
                    break;
                    
                case OP_ADD:
                    matrixAddCUDA(node->inputs[0]->value, node->inputs[1]->value, node->value);
                    break;
                    
                case OP_SUB:
                    matrixSubCUDA(node->inputs[0]->value, node->inputs[1]->value, node->value);
                    break;
                    
                case OP_MUL:
                    matrixMulCUDA(node->inputs[0]->value, node->inputs[1]->value, node->value);
                    break;
                    
                case OP_DIV:
                    matrixDivCUDA(node->inputs[0]->value, node->inputs[1]->value, node->value);
                    break;
                
                case OP_MATMUL:
                    // 实现矩阵乘法前向传播
                    {
                        const int A_rows = node->inputs[0]->value.row;
                        const int A_cols = node->inputs[0]->value.col;
                        const int B_cols = node->inputs[1]->value.col;
                        size_t A_size = A_rows * A_cols * sizeof(float);
                        size_t B_size = A_cols * B_cols * sizeof(float);
                        size_t C_size = A_rows * B_cols * sizeof(float);
                        
                        float *d_A, *d_B, *d_C;
                        CHECK_CUDA_ERROR(hipMalloc(&d_A, A_size));
                        CHECK_CUDA_ERROR(hipMalloc(&d_B, B_size));
                        CHECK_CUDA_ERROR(hipMalloc(&d_C, C_size));
                        
                        CHECK_CUDA_ERROR(hipMemcpy(d_A, node->inputs[0]->value.data, A_size, hipMemcpyHostToDevice));
                        CHECK_CUDA_ERROR(hipMemcpy(d_B, node->inputs[1]->value.data, B_size, hipMemcpyHostToDevice));
                        
                        dim3 blockSize(16, 16);
                        dim3 gridSize((B_cols + blockSize.x - 1) / blockSize.x,
                                      (A_rows + blockSize.y - 1) / blockSize.y);
                        
                        matmulForwardKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, A_rows, A_cols, B_cols);
                        
                        CHECK_CUDA_ERROR(hipGetLastError());
                        CHECK_CUDA_ERROR(hipMemcpy(node->value.data, d_C, C_size, hipMemcpyDeviceToHost));
                        
                        CHECK_CUDA_ERROR(hipFree(d_A));
                        CHECK_CUDA_ERROR(hipFree(d_B));
                        CHECK_CUDA_ERROR(hipFree(d_C));
                    }
                    break;
                    
                case OP_RELU:
                    // 实现ReLU前向传播
                    {
                        const int size = node->value.row * node->value.col;
                        const size_t bytes = size * sizeof(float);
                        
                        float *d_input, *d_output;
                        CHECK_CUDA_ERROR(hipMalloc(&d_input, bytes));
                        CHECK_CUDA_ERROR(hipMalloc(&d_output, bytes));
                        
                        CHECK_CUDA_ERROR(hipMemcpy(d_input, node->inputs[0]->value.data, bytes, hipMemcpyHostToDevice));
                        
                        int blockSize = 256;
                        int numBlocks = (size + blockSize - 1) / blockSize;
                        
                        reluForwardKernel<<<numBlocks, blockSize>>>(d_input, d_output, size);
                        
                        CHECK_CUDA_ERROR(hipGetLastError());
                        CHECK_CUDA_ERROR(hipMemcpy(node->value.data, d_output, bytes, hipMemcpyDeviceToHost));
                        
                        CHECK_CUDA_ERROR(hipFree(d_input));
                        CHECK_CUDA_ERROR(hipFree(d_output));
                    }
                    break;
                    
                case OP_SIGMOID:
                    // 实现Sigmoid前向传播
                    {
                        const int size = node->value.row * node->value.col;
                        const size_t bytes = size * sizeof(float);
                        
                        float *d_input, *d_output;
                        CHECK_CUDA_ERROR(hipMalloc(&d_input, bytes));
                        CHECK_CUDA_ERROR(hipMalloc(&d_output, bytes));
                        
                        CHECK_CUDA_ERROR(hipMemcpy(d_input, node->inputs[0]->value.data, bytes, hipMemcpyHostToDevice));
                        
                        int blockSize = 256;
                        int numBlocks = (size + blockSize - 1) / blockSize;
                        
                        sigmoidForwardKernel<<<numBlocks, blockSize>>>(d_input, d_output, size);
                        
                        CHECK_CUDA_ERROR(hipGetLastError());
                        CHECK_CUDA_ERROR(hipMemcpy(node->value.data, d_output, bytes, hipMemcpyDeviceToHost));
                        
                        CHECK_CUDA_ERROR(hipFree(d_input));
                        CHECK_CUDA_ERROR(hipFree(d_output));
                    }
                    break;
                
                // 其他操作...
                default:
                    std::cerr << "Unsupported operation in forward pass." << std::endl;
                    break;
            }
        }
    }
    
    // 反向传播
    void backward(ComputeNode* outputNode) {
        // 重置所有节点的梯度
        for (ComputeNode* node : nodes) {
            node->zeroGrad();
        }
        
        // 输出节点的梯度初始化为1
        std::fill(outputNode->grad.data, outputNode->grad.data + outputNode->grad.row * outputNode->grad.col, 1.0f);
        
        // 从输出节点开始反向传播
        for (int i = nodes.size() - 1; i >= 0; --i) {
            ComputeNode* node = nodes[i];
            
            switch (node->op) {
                case OP_INPUT:
                    // 输入节点是梯度累积的终点
                    break;
                    
                case OP_ADD:
                    // 加法反向传播
                    addBackwardCUDA(node->grad, node->inputs[0]->grad, node->inputs[1]->grad);
                    break;
                    
                case OP_SUB:
                    // 减法反向传播
                    subBackwardCUDA(node->grad, node->inputs[0]->grad, node->inputs[1]->grad);
                    break;
                    
                case OP_MUL:
                    // 乘法反向传播
                    mulBackwardCUDA(node->grad, node->inputs[0]->value, node->inputs[1]->value, 
                                  node->inputs[0]->grad, node->inputs[1]->grad);
                    break;
                    
                case OP_DIV:
                    // 除法反向传播
                    divBackwardCUDA(node->grad, node->inputs[0]->value, node->inputs[1]->value, 
                                  node->inputs[0]->grad, node->inputs[1]->grad);
                    break;
                
                case OP_MATMUL:
                    // 矩阵乘法反向传播
                    {
                        const int A_rows = node->inputs[0]->value.row;
                        const int A_cols = node->inputs[0]->value.col;
                        const int B_rows = node->inputs[1]->value.row;
                        const int B_cols = node->inputs[1]->value.col;
                        const int outGrad_rows = node->grad.row;
                        const int outGrad_cols = node->grad.col;
                        
                        // A的梯度计算
                        float *d_outGrad, *d_B, *d_inGradA;
                        size_t outGrad_size = outGrad_rows * outGrad_cols * sizeof(float);
                        size_t B_size = B_rows * B_cols * sizeof(float);
                        size_t inGradA_size = A_rows * A_cols * sizeof(float);
                        
                        CHECK_CUDA_ERROR(hipMalloc(&d_outGrad, outGrad_size));
                        CHECK_CUDA_ERROR(hipMalloc(&d_B, B_size));
                        CHECK_CUDA_ERROR(hipMalloc(&d_inGradA, inGradA_size));
                        
                        CHECK_CUDA_ERROR(hipMemcpy(d_outGrad, node->grad.data, outGrad_size, hipMemcpyHostToDevice));
                        CHECK_CUDA_ERROR(hipMemcpy(d_B, node->inputs[1]->value.data, B_size, hipMemcpyHostToDevice));
                        
                        dim3 blockSizeA(16, 16);
                        dim3 gridSizeA((A_cols + blockSizeA.x - 1) / blockSizeA.x,
                                      (A_rows + blockSizeA.y - 1) / blockSizeA.y);
                        
                        matmulBackwardAKernel<<<gridSizeA, blockSizeA>>>(d_outGrad, d_B, d_inGradA, 
                                                                       outGrad_rows, B_rows, B_cols);
                        
                        CHECK_CUDA_ERROR(hipGetLastError());
                        CHECK_CUDA_ERROR(hipMemcpy(node->inputs[0]->grad.data, d_inGradA, inGradA_size, hipMemcpyDeviceToHost));
                        
                        CHECK_CUDA_ERROR(hipFree(d_outGrad));
                        CHECK_CUDA_ERROR(hipFree(d_B));
                        CHECK_CUDA_ERROR(hipFree(d_inGradA));
                        
                        // B的梯度计算
                        float *d_A, *d_inGradB;
                        size_t A_size = A_rows * A_cols * sizeof(float);
                        size_t inGradB_size = B_rows * B_cols * sizeof(float);
                        
                        CHECK_CUDA_ERROR(hipMalloc(&d_outGrad, outGrad_size));
                        CHECK_CUDA_ERROR(hipMalloc(&d_A, A_size));
                        CHECK_CUDA_ERROR(hipMalloc(&d_inGradB, inGradB_size));
                        
                        CHECK_CUDA_ERROR(hipMemcpy(d_outGrad, node->grad.data, outGrad_size, hipMemcpyHostToDevice));
                        CHECK_CUDA_ERROR(hipMemcpy(d_A, node->inputs[0]->value.data, A_size, hipMemcpyHostToDevice));
                        
                        dim3 blockSizeB(16, 16);
                        dim3 gridSizeB((B_cols + blockSizeB.x - 1) / blockSizeB.x,
                                      (B_rows + blockSizeB.y - 1) / blockSizeB.y);
                        
                        matmulBackwardBKernel<<<gridSizeB, blockSizeB>>>(d_A, d_outGrad, d_inGradB, 
                                                                       A_rows, A_cols, outGrad_cols);
                        
                        CHECK_CUDA_ERROR(hipGetLastError());
                        CHECK_CUDA_ERROR(hipMemcpy(node->inputs[1]->grad.data, d_inGradB, inGradB_size, hipMemcpyDeviceToHost));
                        
                        CHECK_CUDA_ERROR(hipFree(d_outGrad));
                        CHECK_CUDA_ERROR(hipFree(d_A));
                        CHECK_CUDA_ERROR(hipFree(d_inGradB));
                    }
                    break;
                    
                case OP_RELU:
                    // ReLU反向传播
                    {
                        const int size = node->grad.row * node->grad.col;
                        const size_t bytes = size * sizeof(float);
                        
                        float *d_outGrad, *d_input, *d_inGrad;
                        CHECK_CUDA_ERROR(hipMalloc(&d_outGrad, bytes));
                        CHECK_CUDA_ERROR(hipMalloc(&d_input, bytes));
                        CHECK_CUDA_ERROR(hipMalloc(&d_inGrad, bytes));
                        
                        CHECK_CUDA_ERROR(hipMemcpy(d_outGrad, node->grad.data, bytes, hipMemcpyHostToDevice));
                        CHECK_CUDA_ERROR(hipMemcpy(d_input, node->inputs[0]->value.data, bytes, hipMemcpyHostToDevice));
                        
                        int blockSize = 256;
                        int numBlocks = (size + blockSize - 1) / blockSize;
                        
                        reluBackwardKernel<<<numBlocks, blockSize>>>(d_outGrad, d_input, d_inGrad, size);
                        
                        CHECK_CUDA_ERROR(hipGetLastError());
                        CHECK_CUDA_ERROR(hipMemcpy(node->inputs[0]->grad.data, d_inGrad, bytes, hipMemcpyDeviceToHost));
                        
                        CHECK_CUDA_ERROR(hipFree(d_outGrad));
                        CHECK_CUDA_ERROR(hipFree(d_input));
                        CHECK_CUDA_ERROR(hipFree(d_inGrad));
                    }
                    break;
                    
                case OP_SIGMOID:
                    // Sigmoid反向传播
                    {
                        const int size = node->grad.row * node->grad.col;
                        const size_t bytes = size * sizeof(float);
                        
                        float *d_outGrad, *d_output, *d_inGrad;
                        CHECK_CUDA_ERROR(hipMalloc(&d_outGrad, bytes));
                        CHECK_CUDA_ERROR(hipMalloc(&d_output, bytes));
                        CHECK_CUDA_ERROR(hipMalloc(&d_inGrad, bytes));
                        
                        CHECK_CUDA_ERROR(hipMemcpy(d_outGrad, node->grad.data, bytes, hipMemcpyHostToDevice));
                        CHECK_CUDA_ERROR(hipMemcpy(d_output, node->value.data, bytes, hipMemcpyHostToDevice));
                        
                        int blockSize = 256;
                        int numBlocks = (size + blockSize - 1) / blockSize;
                        
                        sigmoidBackwardKernel<<<numBlocks, blockSize>>>(d_outGrad, d_output, d_inGrad, size);
                        
                        CHECK_CUDA_ERROR(hipGetLastError());
                        CHECK_CUDA_ERROR(hipMemcpy(node->inputs[0]->grad.data, d_inGrad, bytes, hipMemcpyDeviceToHost));
                        
                        CHECK_CUDA_ERROR(hipFree(d_outGrad));
                        CHECK_CUDA_ERROR(hipFree(d_output));
                        CHECK_CUDA_ERROR(hipFree(d_inGrad));
                    }
                    break;
                
                // 其他操作...
                default:
                    std::cerr << "Unsupported operation in backward pass." << std::endl;
                    break;
            }
        }
    }
};
// CPU version of matrix multiplication
void matrixMatMulCPU(const Matrix_CU& A, const Matrix_CU& B, Matrix_CU& C) {
    assert(A.col == B.row);
    assert(C.row == A.row && C.col == B.col);
    
    for (int i = 0; i < A.row; ++i) {
        for (int j = 0; j < B.col; ++j) {
            C.data[i * C.col + j] = 0.0f;
            for (int k = 0; k < A.col; ++k) {
                C.data[i * C.col + j] += A.data[i * A.col + k] * B.data[k * B.col + j];
            }
        }
    }
}
int main() {
    // 可配置参数
    int input_rows = 10000;        
    int input_cols = 1000;         
    int output_cols = 500;         
    int batch_size = 32;           
    bool use_relu = true;          
    int print_rows = 5;           
    int print_cols = 5;           
    bool verify_with_cpu = false; 
    hipFree(0);
    std::cout << "====== Configuration ======\n";
    std::cout << "Input matrix: " << input_rows << " x " << input_cols << std::endl;
    std::cout << "Weight matrix: " << input_cols << " x " << output_cols << std::endl;
    std::cout << "Batch size: " << batch_size << std::endl;
    std::cout << "Activation: " << (use_relu ? "ReLU" : "None") << std::endl;
    
    // 创建计算图
    ComputeGraph graph;
    
    ComputeNode* input = graph.addInput(input_rows, input_cols, batch_size);
    ComputeNode* weight = graph.addInput(input_cols, output_cols);
    
    input->value.randomInit();
    weight->value.randomInit();
    
    // 创建CPU版本的矩阵用于验证
    Matrix_CU input_cpu = input->value;
    Matrix_CU weight_cpu = weight->value;
    
    // 记录CUDA计算开始时间
    auto cuda_start = std::chrono::high_resolution_clock::now();
    
    // 构建前向计算图
    ComputeNode* fc1 = graph.addMatMul(input, weight);  // 全连接层
    ComputeNode* output_node = fc1;
    
    // 如果需要，添加ReLU激活
    if (use_relu) {
        output_node = graph.addReLU(fc1);
    }
    
    // 计算前向传播
    graph.forward();
    
    // 反向传播计算梯度
    graph.backward(output_node);
    
    // 记录CUDA计算结束时间
    auto cuda_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> cuda_duration = cuda_end - cuda_start;
    
    // 输出CUDA计算结果
    std::cout << "\n====== CUDA Results ======\n";
    std::cout << "Input first " << print_rows << "x" << print_cols << ":\n";
    input->value.printSubMatrix(print_rows, print_cols);
    
    std::cout << "Weight first " << print_rows << "x" << print_cols << ":\n";
    weight->value.printSubMatrix(std::min(print_rows, weight->value.row), 
                                std::min(print_cols, weight->value.col));
    
    std::cout << "Output first " << print_rows << "x" << print_cols << ":\n";
    output_node->value.printSubMatrix(print_rows, 
                                     std::min(print_cols, output_node->value.col));
    
    std::cout << "Input gradient first " << print_rows << "x" << print_cols << ":\n";
    input->grad.printSubMatrix(print_rows, print_cols);
    
    std::cout << "Weight gradient first " << print_rows << "x" << print_cols << ":\n";
    weight->grad.printSubMatrix(std::min(print_rows, weight->value.row), 
                               std::min(print_cols, weight->value.col));
    
    // 如果需要，使用CPU验证
    if (verify_with_cpu) {
        std::cout << "\n====== CPU Verification ======\n";
        
        // 记录CPU计算开始时间
        auto cpu_start = std::chrono::high_resolution_clock::now();
        
        // CPU版本的矩阵乘法计算 (FC1)
        Matrix_CU fc1_cpu(input_cpu.row, weight_cpu.col);
        matrixMatMulCPU(input_cpu, weight_cpu, fc1_cpu);
        
        // CPU版本输出节点
        Matrix_CU output_cpu = fc1_cpu;
        
        // 如果需要，CPU版本的ReLU计算
        if (use_relu) {
            output_cpu = Matrix_CU(fc1_cpu.row, fc1_cpu.col);
            for (int i = 0; i < fc1_cpu.row * fc1_cpu.col; ++i) {
                output_cpu.data[i] = fc1_cpu.data[i] > 0 ? fc1_cpu.data[i] : 0;
            }
        }
        
        // CPU版本的反向传播
        // 初始化输出梯度为1.0
        Matrix_CU output_grad(output_cpu.row, output_cpu.col);
        for (int i = 0; i < output_grad.row * output_grad.col; ++i) {
            output_grad.data[i] = 1.0f;
        }
        
        // ReLU反向传播
        Matrix_CU fc1_grad = output_grad;
        if (use_relu) {
            fc1_grad = Matrix_CU(fc1_cpu.row, fc1_cpu.col);
            for (int i = 0; i < fc1_cpu.row * fc1_cpu.col; ++i) {
                fc1_grad.data[i] = fc1_cpu.data[i] > 0 ? output_grad.data[i] : 0;
            }
        }
        
        // 矩阵乘法反向传播
        Matrix_CU input_grad_cpu(input_cpu.row, input_cpu.col);
        Matrix_CU weight_grad_cpu(weight_cpu.row, weight_cpu.col);
        
        // 计算输入梯度: input_grad = fc1_grad * weight^T
        Matrix_CU weight_transpose(weight_cpu.col, weight_cpu.row);
        for (int i = 0; i < weight_cpu.row; ++i) {
            for (int j = 0; j < weight_cpu.col; ++j) {
                weight_transpose.data[j * weight_cpu.row + i] = weight_cpu.data[i * weight_cpu.col + j];
            }
        }
        matrixMatMulCPU(fc1_grad, weight_transpose, input_grad_cpu);
        
        // 计算权重梯度: weight_grad = input^T * fc1_grad
        Matrix_CU input_transpose(input_cpu.col, input_cpu.row);
        for (int i = 0; i < input_cpu.row; ++i) {
            for (int j = 0; j < input_cpu.col; ++j) {
                input_transpose.data[j * input_cpu.row + i] = input_cpu.data[i * input_cpu.col + j];
            }
        }
        matrixMatMulCPU(input_transpose, fc1_grad, weight_grad_cpu);
        
        // 记录CPU计算结束时间
        auto cpu_end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> cpu_duration = cpu_end - cpu_start;
        
        // 输出CPU计算结果
        std::cout << "CPU Output first " << print_rows << "x" << print_cols << ":\n";
        output_cpu.printSubMatrix(print_rows, std::min(print_cols, output_cpu.col));
        
        std::cout << "CPU Input gradient first " << print_rows << "x" << print_cols << ":\n";
        input_grad_cpu.printSubMatrix(print_rows, print_cols);
        
        std::cout << "CPU Weight gradient first " << print_rows << "x" << print_cols << ":\n";
        weight_grad_cpu.printSubMatrix(std::min(print_rows, weight_grad_cpu.row), 
                                      std::min(print_cols, weight_grad_cpu.col));
        
        // 验证CUDA和CPU结果的差异
        double output_diff = 0.0;
        double input_grad_diff = 0.0;
        double weight_grad_diff = 0.0;
        
        for (int i = 0; i < output_node->value.row * output_node->value.col; ++i) {
            output_diff += std::abs(output_node->value.data[i] - output_cpu.data[i]);
        }
        output_diff /= (output_node->value.row * output_node->value.col);
        
        for (int i = 0; i < input->grad.row * input->grad.col; ++i) {
            input_grad_diff += std::abs(input->grad.data[i] - input_grad_cpu.data[i]);
        }
        input_grad_diff /= (input->grad.row * input->grad.col);
        
        for (int i = 0; i < weight->grad.row * weight->grad.col; ++i) {
            weight_grad_diff += std::abs(weight->grad.data[i] - weight_grad_cpu.data[i]);
        }
        weight_grad_diff /= (weight->grad.row * weight->grad.col);
        
        // 输出结果比较和性能比较
        std::cout << "\n====== Verification Results ======\n";
        std::cout << "Average output difference: " << output_diff << std::endl;
        std::cout << "Average input gradient difference: " << input_grad_diff << std::endl;
        std::cout << "Average weight gradient difference: " << weight_grad_diff << std::endl;
        
        std::cout << "\n====== Performance Comparison ======\n";
        std::cout << "CUDA computation time: " << cuda_duration.count() << " ms" << std::endl;
        std::cout << "CPU computation time: " << cpu_duration.count() << " ms" << std::endl;
        std::cout << "Speedup: " << cpu_duration.count() / cuda_duration.count() << "x" << std::endl;
    } else {
        std::cout << "\n====== Performance ======\n";
        std::cout << "CUDA computation time: " << cuda_duration.count() << " ms" << std::endl;
    }
    
    return 0;
}