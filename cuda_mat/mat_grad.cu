#include "hip/hip_runtime.h"
#include "matrix_cudadef.cu"
#include <iostream>
#include <cstdlib>
#include <chrono>
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstring>
#include <cmath>
#include <functional>
// 自动微分节点类型枚举
enum OpType {
    OP_INPUT,    // 输入节点
    OP_ADD,      // 加法
    OP_SUB,      // 减法
    OP_MUL,      // 元素级乘法
    OP_DIV,      // 元素级除法
    OP_MATMUL,   // 矩阵乘法
    OP_RELU,     // ReLU激活函数
    OP_SIGMOID,  // Sigmoid激活函数
    OP_TANH,     // Tanh激活函数
    OP_SOFTMAX,  // Softmax函数
    OP_CONV2D    // 2D卷积
};
struct ConvParams {
    int in_channels;
    int out_channels;
    int kernel_height;
    int kernel_width;
    int stride_height;
    int stride_width;
    int pad_height;
    int pad_width;
    int in_height;
    int in_width;
    int out_height;
    int out_width;
    int batch_size;
};
// 计算图节点
struct ComputeNode {
    OpType op;                     // 操作类型
    Matrix_CU value;               // 前向传播值
    Matrix_CU grad;                // 梯度值
    std::vector<ComputeNode*> inputs;  // 输入节点
    int batchSize;                 // 批处理大小
    ConvParams conv_params; 
    ComputeNode(int rows, int cols, int batch = 1) : 
        op(OP_INPUT), 
        value(rows * batch, cols), 
        grad(rows * batch, cols), 
        batchSize(batch) {}
    
    // 防止内存泄漏
    ~ComputeNode() {
        // 析构函数会自动处理value和grad
    }
    
    // 重置梯度
    void zeroGrad() {
        std::memset(grad.data, 0, grad.row * grad.col * sizeof(float));
    }
};

// CUDA核函数：ReLU前向传播
__global__ void reluForwardKernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = fmaxf(0.0f, input[idx]);
    }
}

// CUDA核函数：ReLU反向传播
__global__ void reluBackwardKernel(const float* outGrad, const float* input, float* inGrad, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        inGrad[idx] = (input[idx] > 0.0f) ? outGrad[idx] : 0.0f;
    }
}

// CUDA核函数：Sigmoid前向传播
__global__ void sigmoidForwardKernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = 1.0f / (1.0f + expf(-input[idx]));
    }
}

// CUDA核函数：Sigmoid反向传播
__global__ void sigmoidBackwardKernel(const float* outGrad, const float* output, float* inGrad, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // sigmoid的导数是 sigmoid(x) * (1 - sigmoid(x))
        inGrad[idx] = outGrad[idx] * output[idx] * (1.0f - output[idx]);
    }
}

// CUDA核函数：矩阵乘法前向传播 (C = A * B)
__global__ void matmulForwardKernel(const float* A, const float* B, float* C, 
                                   int A_rows, int A_cols, int B_cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < A_rows && col < B_cols) {
        float sum = 0.0f;
        for (int k = 0; k < A_cols; ++k) {
            sum += A[row * A_cols + k] * B[k * B_cols + col];
        }
        C[row * B_cols + col] = sum;
    }
}

// CUDA核函数：矩阵乘法反向传播 - A的梯度
__global__ void matmulBackwardAKernel(const float* outGrad, const float* B, float* inGradA,
                                     int outGrad_rows, int B_rows, int B_cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < outGrad_rows && col < B_rows) {
        float sum = 0.0f;
        for (int k = 0; k < B_cols; ++k) {
            sum += outGrad[row * B_cols + k] * B[col * B_cols + k];
        }
        inGradA[row * B_rows + col] = sum;
    }
}

// CUDA核函数：矩阵乘法反向传播 - B的梯度
__global__ void matmulBackwardBKernel(const float* A, const float* outGrad, float* inGradB,
                                     int A_rows, int A_cols, int outGrad_cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < A_cols && col < outGrad_cols) {
        float sum = 0.0f;
        for (int k = 0; k < A_rows; ++k) {
            sum += A[k * A_cols + row] * outGrad[k * outGrad_cols + col];
        }
        inGradB[row * outGrad_cols + col] = sum;
    }
}
// CUDA kernel for convolution forward pass
__global__ void conv2dForwardKernel(
    const float* input, const float* kernel, float* output,
    int batch_size, int in_channels, int out_channels,
    int in_height, int in_width, int kernel_height, int kernel_width,
    int out_height, int out_width, int stride_height, int stride_width,
    int pad_height, int pad_width) {
    
    int n = blockIdx.x;  // batch index
    int c = blockIdx.y;  // output channel index
    
    int h = (blockIdx.z / out_width);
    int w = (blockIdx.z % out_width);
    
    if (n >= batch_size || c >= out_channels || h >= out_height || w >= out_width)
        return;
    
    float sum = 0.0f;
    
    for (int ic = 0; ic < in_channels; ic++) {
        for (int kh = 0; kh < kernel_height; kh++) {
            for (int kw = 0; kw < kernel_width; kw++) {
                int ih = h * stride_height + kh - pad_height;
                int iw = w * stride_width + kw - pad_width;
                
                if (ih >= 0 && ih < in_height && iw >= 0 && iw < in_width) {
                    int input_idx = ((n * in_channels + ic) * in_height + ih) * in_width + iw;
                    int kernel_idx = ((c * in_channels + ic) * kernel_height + kh) * kernel_width + kw;
                    
                    sum += input[input_idx] * kernel[kernel_idx];
                }
            }
        }
    }
    
    int output_idx = ((n * out_channels + c) * out_height + h) * out_width + w;
    output[output_idx] = sum;
}
// CUDA kernel for computing gradients with respect to input
__global__ void conv2dBackwardInputKernel(
    const float* outGrad, const float* kernel, float* inGrad,
    int batch_size, int in_channels, int out_channels,
    int in_height, int in_width, int kernel_height, int kernel_width,
    int out_height, int out_width, int stride_height, int stride_width,
    int pad_height, int pad_width) {
    
    int n = blockIdx.x;  // batch index
    int ic = blockIdx.y; // input channel index
    int idx = blockIdx.z;
    int h = idx / in_width;
    int w = idx % in_width;
    
    if (n >= batch_size || ic >= in_channels || h >= in_height || w >= in_width)
        return;
    
    float sum = 0.0f;
    
    for (int oc = 0; oc < out_channels; oc++) {
        for (int kh = 0; kh < kernel_height; kh++) {
            for (int kw = 0; kw < kernel_width; kw++) {
                int oh = (h + pad_height - kh) / stride_height;
                int ow = (w + pad_width - kw) / stride_width;
                
                // Check if the computed output location is valid and if it aligns with stride
                if (oh >= 0 && oh < out_height && ow >= 0 && ow < out_width &&
                    (h + pad_height - kh) % stride_height == 0 &&
                    (w + pad_width - kw) % stride_width == 0) {
                    
                    int out_grad_idx = ((n * out_channels + oc) * out_height + oh) * out_width + ow;
                    int kernel_idx = ((oc * in_channels + ic) * kernel_height + kh) * kernel_width + kw;
                    
                    sum += outGrad[out_grad_idx] * kernel[kernel_idx];
                }
            }
        }
    }
    
    int in_grad_idx = ((n * in_channels + ic) * in_height + h) * in_width + w;
    inGrad[in_grad_idx] += sum;  // Use += for gradient accumulation
}

// CUDA kernel for computing gradients with respect to kernel
__global__ void conv2dBackwardKernelKernel(
    const float* input, const float* outGrad, float* kernelGrad,
    int batch_size, int in_channels, int out_channels,
    int in_height, int in_width, int kernel_height, int kernel_width,
    int out_height, int out_width, int stride_height, int stride_width,
    int pad_height, int pad_width) {
    
    int oc = blockIdx.x;  // output channel index
    int ic = blockIdx.y;  // input channel index
    int idx = blockIdx.z;
    int kh = idx / kernel_width;
    int kw = idx % kernel_width;
    
    if (oc >= out_channels || ic >= in_channels || kh >= kernel_height || kw >= kernel_width)
        return;
    
    float sum = 0.0f;
    
    for (int n = 0; n < batch_size; n++) {
        for (int oh = 0; oh < out_height; oh++) {
            for (int ow = 0; ow < out_width; ow++) {
                int ih = oh * stride_height + kh - pad_height;
                int iw = ow * stride_width + kw - pad_width;
                
                if (ih >= 0 && ih < in_height && iw >= 0 && iw < in_width) {
                    int input_idx = ((n * in_channels + ic) * in_height + ih) * in_width + iw;
                    int out_grad_idx = ((n * out_channels + oc) * out_height + oh) * out_width + ow;
                    
                    sum += input[input_idx] * outGrad[out_grad_idx];
                }
            }
        }
    }
    
    int kernel_grad_idx = ((oc * in_channels + ic) * kernel_height + kh) * kernel_width + kw;
    kernelGrad[kernel_grad_idx] = sum;
}

// 为加法和减法操作修改反向传播实现
void addBackwardCUDA(const Matrix_CU& outGrad, Matrix_CU& inGradA, Matrix_CU& inGradB) {
    assert(outGrad.row == inGradA.row && outGrad.col == inGradA.col);
    assert(outGrad.row == inGradB.row && outGrad.col == inGradB.col);
    
    // 加法的梯度直接复制到两个输入（主机内存到主机内存）
    const int size = outGrad.row * outGrad.col * sizeof(float);
    std::memcpy(inGradA.data, outGrad.data, size);
    std::memcpy(inGradB.data, outGrad.data, size);
}
// 添加一个全局的CUDA核函数，替代lambda函数
__global__ void negGradKernel(const float* outGrad, float* inGrad, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        inGrad[idx] = -outGrad[idx];
    }
}
void subBackwardCUDA(const Matrix_CU& outGrad, Matrix_CU& inGradA, Matrix_CU& inGradB) {
    assert(outGrad.row == inGradA.row && outGrad.col == inGradA.col);
    assert(outGrad.row == inGradB.row && outGrad.col == inGradB.col);
    
    const int rows = outGrad.row;
    const int cols = outGrad.col;
    const int totalElements = rows * cols;
    const size_t size = totalElements * sizeof(float);
    
    // 对于A: 直接复制梯度（主机内存到主机内存）
    std::memcpy(inGradA.data, outGrad.data, size);
    
    // 对于B: 计算负梯度
    float *d_outGrad, *d_inGradB;
    CHECK_CUDA_ERROR(hipMalloc(&d_outGrad, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_inGradB, size));
    
    CHECK_CUDA_ERROR(hipMemcpy(d_outGrad, outGrad.data, size, hipMemcpyHostToDevice));
    
    int blockSize = 256;
    int numBlocks = (totalElements + blockSize - 1) / blockSize;
    
    negGradKernel<<<numBlocks, blockSize>>>(d_outGrad, d_inGradB, totalElements);
    
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipMemcpy(inGradB.data, d_inGradB, size, hipMemcpyDeviceToHost));
    
    CHECK_CUDA_ERROR(hipFree(d_outGrad));
    CHECK_CUDA_ERROR(hipFree(d_inGradB));
}




// 元素级乘法的反向传播CUDA实现
__global__ void mulBackwardKernel(const float* outGrad, const float* A, const float* B, 
                                 float* inGradA, float* inGradB, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // 对于A: ∂L/∂A = ∂L/∂(A*B) * ∂(A*B)/∂A = ∂L/∂(A*B) * B
        inGradA[idx] = outGrad[idx] * B[idx];
        
        // 对于B: ∂L/∂B = ∂L/∂(A*B) * ∂(A*B)/∂B = ∂L/∂(A*B) * A
        inGradB[idx] = outGrad[idx] * A[idx];
    }
}

void mulBackwardCUDA(const Matrix_CU& outGrad, const Matrix_CU& A, const Matrix_CU& B, 
                    Matrix_CU& inGradA, Matrix_CU& inGradB) {
    assert(outGrad.row == A.row && outGrad.col == A.col);
    assert(A.row == B.row && A.col == B.col);
    assert(outGrad.row == inGradA.row && outGrad.col == inGradA.col);
    assert(outGrad.row == inGradB.row && outGrad.col == inGradB.col);
    
    const int rows = outGrad.row;
    const int cols = outGrad.col;
    const int totalElements = rows * cols;
    const size_t size = totalElements * sizeof(float);
    
    float *d_outGrad, *d_A, *d_B, *d_inGradA, *d_inGradB;
    
    CHECK_CUDA_ERROR(hipMalloc(&d_outGrad, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_A, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_B, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_inGradA, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_inGradB, size));
    
    CHECK_CUDA_ERROR(hipMemcpy(d_outGrad, outGrad.data, size, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_A, A.data, size, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_B, B.data, size, hipMemcpyHostToDevice));
    
    int blockSize = 256;
    int numBlocks = (totalElements + blockSize - 1) / blockSize;
    
    mulBackwardKernel<<<numBlocks, blockSize>>>(d_outGrad, d_A, d_B, d_inGradA, d_inGradB, totalElements);
    
    CHECK_CUDA_ERROR(hipGetLastError());
    
    CHECK_CUDA_ERROR(hipMemcpy(inGradA.data, d_inGradA, size, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(inGradB.data, d_inGradB, size, hipMemcpyDeviceToHost));
    
    CHECK_CUDA_ERROR(hipFree(d_outGrad));
    CHECK_CUDA_ERROR(hipFree(d_A));
    CHECK_CUDA_ERROR(hipFree(d_B));
    CHECK_CUDA_ERROR(hipFree(d_inGradA));
    CHECK_CUDA_ERROR(hipFree(d_inGradB));
}

// 元素级除法的反向传播CUDA实现
__global__ void divBackwardKernel(const float* outGrad, const float* A, const float* B, 
                                float* inGradA, float* inGradB, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // 对于A: ∂L/∂A = ∂L/∂(A/B) * ∂(A/B)/∂A = ∂L/∂(A/B) * (1/B)
        inGradA[idx] = (B[idx] != 0.0f) ? (outGrad[idx] / B[idx]) : 0.0f;
        
        // 对于B: ∂L/∂B = ∂L/∂(A/B) * ∂(A/B)/∂B = ∂L/∂(A/B) * (-A/B²)
        inGradB[idx] = (B[idx] != 0.0f) ? (outGrad[idx] * (-A[idx] / (B[idx] * B[idx]))) : 0.0f;
    }
}

void divBackwardCUDA(const Matrix_CU& outGrad, const Matrix_CU& A, const Matrix_CU& B, 
                   Matrix_CU& inGradA, Matrix_CU& inGradB) {
    // 与乘法类似，但导数计算不同
    assert(outGrad.row == A.row && outGrad.col == A.col);
    assert(A.row == B.row && A.col == B.col);
    assert(outGrad.row == inGradA.row && outGrad.col == inGradA.col);
    assert(outGrad.row == inGradB.row && outGrad.col == inGradB.col);
    
    const int rows = outGrad.row;
    const int cols = outGrad.col;
    const int totalElements = rows * cols;
    const size_t size = totalElements * sizeof(float);
    
    float *d_outGrad, *d_A, *d_B, *d_inGradA, *d_inGradB;
    
    CHECK_CUDA_ERROR(hipMalloc(&d_outGrad, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_A, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_B, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_inGradA, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_inGradB, size));
    
    CHECK_CUDA_ERROR(hipMemcpy(d_outGrad, outGrad.data, size, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_A, A.data, size, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_B, B.data, size, hipMemcpyHostToDevice));
    
    int blockSize = 256;
    int numBlocks = (totalElements + blockSize - 1) / blockSize;
    
    divBackwardKernel<<<numBlocks, blockSize>>>(d_outGrad, d_A, d_B, d_inGradA, d_inGradB, totalElements);
    
    CHECK_CUDA_ERROR(hipGetLastError());
    
    CHECK_CUDA_ERROR(hipMemcpy(inGradA.data, d_inGradA, size, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(inGradB.data, d_inGradB, size, hipMemcpyDeviceToHost));
    
    CHECK_CUDA_ERROR(hipFree(d_outGrad));
    CHECK_CUDA_ERROR(hipFree(d_A));
    CHECK_CUDA_ERROR(hipFree(d_B));
    CHECK_CUDA_ERROR(hipFree(d_inGradA));
    CHECK_CUDA_ERROR(hipFree(d_inGradB));
}

// 计算图管理类
class ComputeGraph {
public:
    std::vector<ComputeNode*> nodes;
    
    ~ComputeGraph() {
        for (auto node : nodes) {
            delete node;
        }
    }
    
    // 添加输入节点
    ComputeNode* addInput(int rows, int cols, int batchSize = 1) {
        ComputeNode* node = new ComputeNode(rows, cols, batchSize);
        node->op = OP_INPUT;
        nodes.push_back(node);
        return node;
    }
    
    // 添加加法节点
    ComputeNode* addAdd(ComputeNode* a, ComputeNode* b) {
        assert(a->value.row == b->value.row && a->value.col == b->value.col);
        
        ComputeNode* node = new ComputeNode(a->value.row / a->batchSize, a->value.col, a->batchSize);
        node->op = OP_ADD;
        node->inputs.push_back(a);
        node->inputs.push_back(b);
        nodes.push_back(node);
        return node;
    }
    
    // 添加减法节点
    ComputeNode* addSub(ComputeNode* a, ComputeNode* b) {
        assert(a->value.row == b->value.row && a->value.col == b->value.col);
        
        ComputeNode* node = new ComputeNode(a->value.row / a->batchSize, a->value.col, a->batchSize);
        node->op = OP_SUB;
        node->inputs.push_back(a);
        node->inputs.push_back(b);
        nodes.push_back(node);
        return node;
    }
    
    // 添加元素级乘法节点
    ComputeNode* addMul(ComputeNode* a, ComputeNode* b) {
        assert(a->value.row == b->value.row && a->value.col == b->value.col);
        
        ComputeNode* node = new ComputeNode(a->value.row / a->batchSize, a->value.col, a->batchSize);
        node->op = OP_MUL;
        node->inputs.push_back(a);
        node->inputs.push_back(b);
        nodes.push_back(node);
        return node;
    }
    
    // 添加元素级除法节点
    ComputeNode* addDiv(ComputeNode* a, ComputeNode* b) {
        assert(a->value.row == b->value.row && a->value.col == b->value.col);
        
        ComputeNode* node = new ComputeNode(a->value.row / a->batchSize, a->value.col, a->batchSize);
        node->op = OP_DIV;
        node->inputs.push_back(a);
        node->inputs.push_back(b);
        nodes.push_back(node);
        return node;
    }
    
    // 添加矩阵乘法节点
    ComputeNode* addMatMul(ComputeNode* a, ComputeNode* b) {
        assert(a->value.col == b->value.row);
        
        ComputeNode* node = new ComputeNode(a->value.row / a->batchSize, b->value.col, a->batchSize);
        node->op = OP_MATMUL;
        node->inputs.push_back(a);
        node->inputs.push_back(b);
        nodes.push_back(node);
        return node;
    }
    
    // 添加ReLU激活函数节点
    ComputeNode* addReLU(ComputeNode* input) {
        ComputeNode* node = new ComputeNode(input->value.row / input->batchSize, input->value.col, input->batchSize);
        node->op = OP_RELU;
        node->inputs.push_back(input);
        nodes.push_back(node);
        return node;
    }
    
    // 添加Sigmoid激活函数节点
    ComputeNode* addSigmoid(ComputeNode* input) {
        ComputeNode* node = new ComputeNode(input->value.row / input->batchSize, input->value.col, input->batchSize);
        node->op = OP_SIGMOID;
        node->inputs.push_back(input);
        nodes.push_back(node);
        return node;
    }
    ComputeNode* addConv2D(ComputeNode* input, ComputeNode* kernel, 
                         int in_height, int in_width, int in_channels,
                         int stride_height = 1, int stride_width = 1, 
                         int pad_height = 0, int pad_width = 0) {
        
        int batch_size = input->batchSize;
        int out_channels = kernel->value.row;
        
        // Calculate kernel dimensions
        int kernel_height, kernel_width;
        kernel_height = kernel_width = (int)sqrt(kernel->value.col / in_channels);
        
        // Calculate output dimensions
        int out_height = ((in_height + 2 * pad_height - kernel_height) / stride_height) + 1;
        int out_width = ((in_width + 2 * pad_width - kernel_width) / stride_width) + 1;
        
        // Create output node - result will be [batch_size * out_channels, out_height * out_width]
        ComputeNode* node = new ComputeNode(out_channels, out_height * out_width, batch_size);
        node->op = OP_CONV2D;
        node->inputs.push_back(input);
        node->inputs.push_back(kernel);
        
        // Store convolution parameters
        node->conv_params.in_channels = in_channels;
        node->conv_params.out_channels = out_channels;
        node->conv_params.kernel_height = kernel_height;
        node->conv_params.kernel_width = kernel_width;
        node->conv_params.stride_height = stride_height;
        node->conv_params.stride_width = stride_width;
        node->conv_params.pad_height = pad_height;
        node->conv_params.pad_width = pad_width;
        node->conv_params.in_height = in_height;
        node->conv_params.in_width = in_width;
        node->conv_params.out_height = out_height;
        node->conv_params.out_width = out_width;
        node->conv_params.batch_size = batch_size;
        
        nodes.push_back(node);
        return node;
    }
    // 前向传播
    void forward() {
        for (ComputeNode* node : nodes) {
            switch (node->op) {
                case OP_CONV2D:
                    // Implement convolution forward pass
                    {
                        const ConvParams& params = node->conv_params;
                        
                        // Reshape input from [batch_size * in_channels, in_height * in_width] 
                        // to [batch_size, in_channels, in_height, in_width]
                        std::vector<float> input_reshaped(params.batch_size * params.in_channels * 
                                                        params.in_height * params.in_width);
                        
                        // Reshape kernel from [out_channels, in_channels * kernel_height * kernel_width]
                        // to [out_channels, in_channels, kernel_height, kernel_width]
                        std::vector<float> kernel_reshaped(params.out_channels * params.in_channels * 
                                                         params.kernel_height * params.kernel_width);
                        
                        // Manual reshape for input
                        for (int n = 0; n < params.batch_size; n++) {
                            for (int c = 0; c < params.in_channels; c++) {
                                for (int h = 0; h < params.in_height; h++) {
                                    for (int w = 0; w < params.in_width; w++) {
                                        int flat_idx = (n * params.in_channels + c) * (params.in_height * params.in_width) + 
                                                     (h * params.in_width + w);
                                        int tensor_idx = ((n * params.in_channels + c) * params.in_height + h) * 
                                                       params.in_width + w;
                                        input_reshaped[tensor_idx] = node->inputs[0]->value.data[flat_idx];
                                    }
                                }
                            }
                        }
                        
                        // Manual reshape for kernel
                        for (int oc = 0; oc < params.out_channels; oc++) {
                            for (int ic = 0; ic < params.in_channels; ic++) {
                                for (int kh = 0; kh < params.kernel_height; kh++) {
                                    for (int kw = 0; kw < params.kernel_width; kw++) {
                                        int flat_idx = oc * (params.in_channels * params.kernel_height * params.kernel_width) + 
                                                     (ic * params.kernel_height * params.kernel_width + 
                                                      kh * params.kernel_width + kw);
                                        int tensor_idx = ((oc * params.in_channels + ic) * params.kernel_height + kh) * 
                                                       params.kernel_width + kw;
                                        kernel_reshaped[tensor_idx] = node->inputs[1]->value.data[flat_idx];
                                    }
                                }
                            }
                        }
                        
                        // Output tensor
                        std::vector<float> output_tensor(params.batch_size * params.out_channels * 
                                                      params.out_height * params.out_width, 0.0f);
                        
                        // Allocate device memory
                        float *d_input, *d_kernel, *d_output;
                        size_t input_size = input_reshaped.size() * sizeof(float);
                        size_t kernel_size = kernel_reshaped.size() * sizeof(float);
                        size_t output_size = output_tensor.size() * sizeof(float);
                        
                        CHECK_CUDA_ERROR(hipMalloc(&d_input, input_size));
                        CHECK_CUDA_ERROR(hipMalloc(&d_kernel, kernel_size));
                        CHECK_CUDA_ERROR(hipMalloc(&d_output, output_size));
                        
                        CHECK_CUDA_ERROR(hipMemcpy(d_input, input_reshaped.data(), input_size, hipMemcpyHostToDevice));
                        CHECK_CUDA_ERROR(hipMemcpy(d_kernel, kernel_reshaped.data(), kernel_size, hipMemcpyHostToDevice));
                        
                        // Launch kernel
                        dim3 gridSize(params.batch_size, params.out_channels, params.out_height * params.out_width);
                        dim3 blockSize(1, 1, 1);  // Simple configuration
                        
                        conv2dForwardKernel<<<gridSize, blockSize>>>(
                            d_input, d_kernel, d_output,
                            params.batch_size, params.in_channels, params.out_channels,
                            params.in_height, params.in_width, params.kernel_height, params.kernel_width,
                            params.out_height, params.out_width, params.stride_height, params.stride_width,
                            params.pad_height, params.pad_width
                        );
                        
                        CHECK_CUDA_ERROR(hipGetLastError());
                        CHECK_CUDA_ERROR(hipMemcpy(output_tensor.data(), d_output, output_size, hipMemcpyDeviceToHost));
                        
                        // Reshape output back to matrix format
                        for (int n = 0; n < params.batch_size; n++) {
                            for (int c = 0; c < params.out_channels; c++) {
                                for (int h = 0; h < params.out_height; h++) {
                                    for (int w = 0; w < params.out_width; w++) {
                                        int tensor_idx = ((n * params.out_channels + c) * params.out_height + h) * 
                                                       params.out_width + w;
                                        int flat_idx = (n * params.out_channels + c) * (params.out_height * params.out_width) + 
                                                     (h * params.out_width + w);
                                        node->value.data[flat_idx] = output_tensor[tensor_idx];
                                    }
                                }
                            }
                        }
                        
                        CHECK_CUDA_ERROR(hipFree(d_input));
                        CHECK_CUDA_ERROR(hipFree(d_kernel));
                        CHECK_CUDA_ERROR(hipFree(d_output));
                    }
                    break;
                case OP_INPUT:
                    // 输入节点不需要计算
                    break;
                    
                case OP_ADD:
                    matrixAddCUDA(node->inputs[0]->value, node->inputs[1]->value, node->value);
                    break;
                    
                case OP_SUB:
                    matrixSubCUDA(node->inputs[0]->value, node->inputs[1]->value, node->value);
                    break;
                    
                case OP_MUL:
                    matrixMulCUDA(node->inputs[0]->value, node->inputs[1]->value, node->value);
                    break;
                    
                case OP_DIV:
                    matrixDivCUDA(node->inputs[0]->value, node->inputs[1]->value, node->value);
                    break;
                
                case OP_MATMUL:
                    // 实现矩阵乘法前向传播
                    {
                        const int A_rows = node->inputs[0]->value.row;
                        const int A_cols = node->inputs[0]->value.col;
                        const int B_cols = node->inputs[1]->value.col;
                        size_t A_size = A_rows * A_cols * sizeof(float);
                        size_t B_size = A_cols * B_cols * sizeof(float);
                        size_t C_size = A_rows * B_cols * sizeof(float);
                        
                        float *d_A, *d_B, *d_C;
                        CHECK_CUDA_ERROR(hipMalloc(&d_A, A_size));
                        CHECK_CUDA_ERROR(hipMalloc(&d_B, B_size));
                        CHECK_CUDA_ERROR(hipMalloc(&d_C, C_size));
                        
                        CHECK_CUDA_ERROR(hipMemcpy(d_A, node->inputs[0]->value.data, A_size, hipMemcpyHostToDevice));
                        CHECK_CUDA_ERROR(hipMemcpy(d_B, node->inputs[1]->value.data, B_size, hipMemcpyHostToDevice));
                        
                        dim3 blockSize(16, 16);
                        dim3 gridSize((B_cols + blockSize.x - 1) / blockSize.x,
                                      (A_rows + blockSize.y - 1) / blockSize.y);
                        
                        matmulForwardKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, A_rows, A_cols, B_cols);
                        
                        CHECK_CUDA_ERROR(hipGetLastError());
                        CHECK_CUDA_ERROR(hipMemcpy(node->value.data, d_C, C_size, hipMemcpyDeviceToHost));
                        
                        CHECK_CUDA_ERROR(hipFree(d_A));
                        CHECK_CUDA_ERROR(hipFree(d_B));
                        CHECK_CUDA_ERROR(hipFree(d_C));
                    }
                    break;
                    
                case OP_RELU:
                    // 实现ReLU前向传播
                    {
                        const int size = node->value.row * node->value.col;
                        const size_t bytes = size * sizeof(float);
                        
                        float *d_input, *d_output;
                        CHECK_CUDA_ERROR(hipMalloc(&d_input, bytes));
                        CHECK_CUDA_ERROR(hipMalloc(&d_output, bytes));
                        
                        CHECK_CUDA_ERROR(hipMemcpy(d_input, node->inputs[0]->value.data, bytes, hipMemcpyHostToDevice));
                        
                        int blockSize = 256;
                        int numBlocks = (size + blockSize - 1) / blockSize;
                        
                        reluForwardKernel<<<numBlocks, blockSize>>>(d_input, d_output, size);
                        
                        CHECK_CUDA_ERROR(hipGetLastError());
                        CHECK_CUDA_ERROR(hipMemcpy(node->value.data, d_output, bytes, hipMemcpyDeviceToHost));
                        
                        CHECK_CUDA_ERROR(hipFree(d_input));
                        CHECK_CUDA_ERROR(hipFree(d_output));
                    }
                    break;
                    
                case OP_SIGMOID:
                    // 实现Sigmoid前向传播
                    {
                        const int size = node->value.row * node->value.col;
                        const size_t bytes = size * sizeof(float);
                        
                        float *d_input, *d_output;
                        CHECK_CUDA_ERROR(hipMalloc(&d_input, bytes));
                        CHECK_CUDA_ERROR(hipMalloc(&d_output, bytes));
                        
                        CHECK_CUDA_ERROR(hipMemcpy(d_input, node->inputs[0]->value.data, bytes, hipMemcpyHostToDevice));
                        
                        int blockSize = 256;
                        int numBlocks = (size + blockSize - 1) / blockSize;
                        
                        sigmoidForwardKernel<<<numBlocks, blockSize>>>(d_input, d_output, size);
                        
                        CHECK_CUDA_ERROR(hipGetLastError());
                        CHECK_CUDA_ERROR(hipMemcpy(node->value.data, d_output, bytes, hipMemcpyDeviceToHost));
                        
                        CHECK_CUDA_ERROR(hipFree(d_input));
                        CHECK_CUDA_ERROR(hipFree(d_output));
                    }
                    break;
                
                // 其他操作...
                default:
                    std::cerr << "Unsupported operation in forward pass." << std::endl;
                    break;
            }
        }
    }
    
    // 反向传播
    void backward(ComputeNode* outputNode) {
        // 重置所有节点的梯度
        for (ComputeNode* node : nodes) {
            node->zeroGrad();
        }
        
        // 输出节点的梯度初始化为1
        std::fill(outputNode->grad.data, outputNode->grad.data + outputNode->grad.row * outputNode->grad.col, 1.0f);
        
        // 从输出节点开始反向传播
        for (int i = nodes.size() - 1; i >= 0; --i) {
            ComputeNode* node = nodes[i];
            switch (node->op) {
                case OP_CONV2D:
                    {
                        const ConvParams& params = node->conv_params;
                        
                        // Reshape gradients and values for convolution backprop
                        std::vector<float> outGrad_reshaped(params.batch_size * params.out_channels * 
                                                         params.out_height * params.out_width);
                        std::vector<float> input_reshaped(params.batch_size * params.in_channels * 
                                                       params.in_height * params.in_width);
                        std::vector<float> kernel_reshaped(params.out_channels * params.in_channels * 
                                                        params.kernel_height * params.kernel_width);
                        
                        // Reshape output gradient
                        for (int n = 0; n < params.batch_size; n++) {
                            for (int c = 0; c < params.out_channels; c++) {
                                for (int h = 0; h < params.out_height; h++) {
                                    for (int w = 0; w < params.out_width; w++) {
                                        int flat_idx = (n * params.out_channels + c) * (params.out_height * params.out_width) + 
                                                     (h * params.out_width + w);
                                        int tensor_idx = ((n * params.out_channels + c) * params.out_height + h) * 
                                                       params.out_width + w;
                                        outGrad_reshaped[tensor_idx] = node->grad.data[flat_idx];
                                    }
                                }
                            }
                        }
                        
                        // Reshape input
                        for (int n = 0; n < params.batch_size; n++) {
                            for (int c = 0; c < params.in_channels; c++) {
                                for (int h = 0; h < params.in_height; h++) {
                                    for (int w = 0; w < params.in_width; w++) {
                                        int flat_idx = (n * params.in_channels + c) * (params.in_height * params.in_width) + 
                                                     (h * params.in_width + w);
                                        int tensor_idx = ((n * params.in_channels + c) * params.in_height + h) * 
                                                       params.in_width + w;
                                        input_reshaped[tensor_idx] = node->inputs[0]->value.data[flat_idx];
                                    }
                                }
                            }
                        }
                        
                        // Reshape kernel
                        for (int oc = 0; oc < params.out_channels; oc++) {
                            for (int ic = 0; ic < params.in_channels; ic++) {
                                for (int kh = 0; kh < params.kernel_height; kh++) {
                                    for (int kw = 0; kw < params.kernel_width; kw++) {
                                        int flat_idx = oc * (params.in_channels * params.kernel_height * params.kernel_width) + 
                                                     (ic * params.kernel_height * params.kernel_width + 
                                                      kh * params.kernel_width + kw);
                                        int tensor_idx = ((oc * params.in_channels + ic) * params.kernel_height + kh) * 
                                                       params.kernel_width + kw;
                                        kernel_reshaped[tensor_idx] = node->inputs[1]->value.data[flat_idx];
                                    }
                                }
                            }
                        }
                        
                        // Prepare gradient outputs
                        std::vector<float> inGrad_reshaped(params.batch_size * params.in_channels * 
                                                        params.in_height * params.in_width, 0.0f);
                        std::vector<float> kernelGrad_reshaped(params.out_channels * params.in_channels * 
                                                            params.kernel_height * params.kernel_width, 0.0f);
                        
                        // Allocate device memory
                        float *d_outGrad, *d_input, *d_kernel, *d_inGrad, *d_kernelGrad;
                        size_t outGrad_size = outGrad_reshaped.size() * sizeof(float);
                        size_t input_size = input_reshaped.size() * sizeof(float);
                        size_t kernel_size = kernel_reshaped.size() * sizeof(float);
                        size_t inGrad_size = inGrad_reshaped.size() * sizeof(float);
                        size_t kernelGrad_size = kernelGrad_reshaped.size() * sizeof(float);
                        
                        CHECK_CUDA_ERROR(hipMalloc(&d_outGrad, outGrad_size));
                        CHECK_CUDA_ERROR(hipMalloc(&d_input, input_size));
                        CHECK_CUDA_ERROR(hipMalloc(&d_kernel, kernel_size));
                        CHECK_CUDA_ERROR(hipMalloc(&d_inGrad, inGrad_size));
                        CHECK_CUDA_ERROR(hipMalloc(&d_kernelGrad, kernelGrad_size));
                        
                        CHECK_CUDA_ERROR(hipMemcpy(d_outGrad, outGrad_reshaped.data(), outGrad_size, hipMemcpyHostToDevice));
                        CHECK_CUDA_ERROR(hipMemcpy(d_input, input_reshaped.data(), input_size, hipMemcpyHostToDevice));
                        CHECK_CUDA_ERROR(hipMemcpy(d_kernel, kernel_reshaped.data(), kernel_size, hipMemcpyHostToDevice));
                        CHECK_CUDA_ERROR(hipMemcpy(d_inGrad, inGrad_reshaped.data(), inGrad_size, hipMemcpyHostToDevice));
                        
                        // Launch input gradient kernel
                        dim3 gridSizeInput(params.batch_size, params.in_channels, params.in_height * params.in_width);
                        dim3 blockSizeInput(1, 1, 1);
                        
                        conv2dBackwardInputKernel<<<gridSizeInput, blockSizeInput>>>(
                            d_outGrad, d_kernel, d_inGrad,
                            params.batch_size, params.in_channels, params.out_channels,
                            params.in_height, params.in_width, params.kernel_height, params.kernel_width,
                            params.out_height, params.out_width, params.stride_height, params.stride_width,
                            params.pad_height, params.pad_width
                        );
                        
                        // Launch kernel gradient kernel
                        dim3 gridSizeKernel(params.out_channels, params.in_channels, 
                                          params.kernel_height * params.kernel_width);
                        dim3 blockSizeKernel(1, 1, 1);
                        
                        conv2dBackwardKernelKernel<<<gridSizeKernel, blockSizeKernel>>>(
                            d_input, d_outGrad, d_kernelGrad,
                            params.batch_size, params.in_channels, params.out_channels,
                            params.in_height, params.in_width, params.kernel_height, params.kernel_width,
                            params.out_height, params.out_width, params.stride_height, params.stride_width,
                            params.pad_height, params.pad_width
                        );
                        
                        CHECK_CUDA_ERROR(hipGetLastError());
                        
                        CHECK_CUDA_ERROR(hipMemcpy(inGrad_reshaped.data(), d_inGrad, inGrad_size, hipMemcpyDeviceToHost));
                        CHECK_CUDA_ERROR(hipMemcpy(kernelGrad_reshaped.data(), d_kernelGrad, kernelGrad_size, hipMemcpyDeviceToHost));
                        
                        // Reshape gradients back to matrix format
                        for (int n = 0; n < params.batch_size; n++) {
                            for (int c = 0; c < params.in_channels; c++) {
                                for (int h = 0; h < params.in_height; h++) {
                                    for (int w = 0; w < params.in_width; w++) {
                                        int tensor_idx = ((n * params.in_channels + c) * params.in_height + h) * 
                                                       params.in_width + w;
                                        int flat_idx = (n * params.in_channels + c) * (params.in_height * params.in_width) + 
                                                     (h * params.in_width + w);
                                        node->inputs[0]->grad.data[flat_idx] += inGrad_reshaped[tensor_idx];
                                    }
                                }
                            }
                        }
                        
                        for (int oc = 0; oc < params.out_channels; oc++) {
                            for (int ic = 0; ic < params.in_channels; ic++) {
                                for (int kh = 0; kh < params.kernel_height; kh++) {
                                    for (int kw = 0; kw < params.kernel_width; kw++) {
                                        int tensor_idx = ((oc * params.in_channels + ic) * params.kernel_height + kh) * 
                                                       params.kernel_width + kw;
                                        int flat_idx = oc * (params.in_channels * params.kernel_height * params.kernel_width) + 
                                                     (ic * params.kernel_height * params.kernel_width + 
                                                      kh * params.kernel_width + kw);
                                        node->inputs[1]->grad.data[flat_idx] += kernelGrad_reshaped[tensor_idx];
                                    }
                                }
                            }
                        }
                        
                        CHECK_CUDA_ERROR(hipFree(d_outGrad));
                        CHECK_CUDA_ERROR(hipFree(d_input));
                        CHECK_CUDA_ERROR(hipFree(d_kernel));
                        CHECK_CUDA_ERROR(hipFree(d_inGrad));
                        CHECK_CUDA_ERROR(hipFree(d_kernelGrad));
                    }
                    break;
                case OP_INPUT:
                    // 输入节点是梯度累积的终点
                    break;
                    
                case OP_ADD:
                    // 加法反向传播
                    addBackwardCUDA(node->grad, node->inputs[0]->grad, node->inputs[1]->grad);
                    break;
                    
                case OP_SUB:
                    // 减法反向传播
                    subBackwardCUDA(node->grad, node->inputs[0]->grad, node->inputs[1]->grad);
                    break;
                    
                case OP_MUL:
                    // 乘法反向传播
                    mulBackwardCUDA(node->grad, node->inputs[0]->value, node->inputs[1]->value, 
                                  node->inputs[0]->grad, node->inputs[1]->grad);
                    break;
                    
                case OP_DIV:
                    // 除法反向传播
                    divBackwardCUDA(node->grad, node->inputs[0]->value, node->inputs[1]->value, 
                                  node->inputs[0]->grad, node->inputs[1]->grad);
                    break;
                
                case OP_MATMUL:
                    // 矩阵乘法反向传播
                    {
                        const int A_rows = node->inputs[0]->value.row;
                        const int A_cols = node->inputs[0]->value.col;
                        const int B_rows = node->inputs[1]->value.row;
                        const int B_cols = node->inputs[1]->value.col;
                        const int outGrad_rows = node->grad.row;
                        const int outGrad_cols = node->grad.col;
                        
                        // A的梯度计算
                        float *d_outGrad, *d_B, *d_inGradA;
                        size_t outGrad_size = outGrad_rows * outGrad_cols * sizeof(float);
                        size_t B_size = B_rows * B_cols * sizeof(float);
                        size_t inGradA_size = A_rows * A_cols * sizeof(float);
                        
                        CHECK_CUDA_ERROR(hipMalloc(&d_outGrad, outGrad_size));
                        CHECK_CUDA_ERROR(hipMalloc(&d_B, B_size));
                        CHECK_CUDA_ERROR(hipMalloc(&d_inGradA, inGradA_size));
                        
                        CHECK_CUDA_ERROR(hipMemcpy(d_outGrad, node->grad.data, outGrad_size, hipMemcpyHostToDevice));
                        CHECK_CUDA_ERROR(hipMemcpy(d_B, node->inputs[1]->value.data, B_size, hipMemcpyHostToDevice));
                        
                        dim3 blockSizeA(16, 16);
                        dim3 gridSizeA((A_cols + blockSizeA.x - 1) / blockSizeA.x,
                                      (A_rows + blockSizeA.y - 1) / blockSizeA.y);
                        
                        matmulBackwardAKernel<<<gridSizeA, blockSizeA>>>(d_outGrad, d_B, d_inGradA, 
                                                                       outGrad_rows, B_rows, B_cols);
                        
                        CHECK_CUDA_ERROR(hipGetLastError());
                        CHECK_CUDA_ERROR(hipMemcpy(node->inputs[0]->grad.data, d_inGradA, inGradA_size, hipMemcpyDeviceToHost));
                        
                        CHECK_CUDA_ERROR(hipFree(d_outGrad));
                        CHECK_CUDA_ERROR(hipFree(d_B));
                        CHECK_CUDA_ERROR(hipFree(d_inGradA));
                        
                        // B的梯度计算
                        float *d_A, *d_inGradB;
                        size_t A_size = A_rows * A_cols * sizeof(float);
                        size_t inGradB_size = B_rows * B_cols * sizeof(float);
                        
                        CHECK_CUDA_ERROR(hipMalloc(&d_outGrad, outGrad_size));
                        CHECK_CUDA_ERROR(hipMalloc(&d_A, A_size));
                        CHECK_CUDA_ERROR(hipMalloc(&d_inGradB, inGradB_size));
                        
                        CHECK_CUDA_ERROR(hipMemcpy(d_outGrad, node->grad.data, outGrad_size, hipMemcpyHostToDevice));
                        CHECK_CUDA_ERROR(hipMemcpy(d_A, node->inputs[0]->value.data, A_size, hipMemcpyHostToDevice));
                        
                        dim3 blockSizeB(16, 16);
                        dim3 gridSizeB((B_cols + blockSizeB.x - 1) / blockSizeB.x,
                                      (B_rows + blockSizeB.y - 1) / blockSizeB.y);
                        
                        matmulBackwardBKernel<<<gridSizeB, blockSizeB>>>(d_A, d_outGrad, d_inGradB, 
                                                                       A_rows, A_cols, outGrad_cols);
                        
                        CHECK_CUDA_ERROR(hipGetLastError());
                        CHECK_CUDA_ERROR(hipMemcpy(node->inputs[1]->grad.data, d_inGradB, inGradB_size, hipMemcpyDeviceToHost));
                        
                        CHECK_CUDA_ERROR(hipFree(d_outGrad));
                        CHECK_CUDA_ERROR(hipFree(d_A));
                        CHECK_CUDA_ERROR(hipFree(d_inGradB));
                    }
                    break;
                    
                case OP_RELU:
                    // ReLU反向传播
                    {
                        const int size = node->grad.row * node->grad.col;
                        const size_t bytes = size * sizeof(float);
                        
                        float *d_outGrad, *d_input, *d_inGrad;
                        CHECK_CUDA_ERROR(hipMalloc(&d_outGrad, bytes));
                        CHECK_CUDA_ERROR(hipMalloc(&d_input, bytes));
                        CHECK_CUDA_ERROR(hipMalloc(&d_inGrad, bytes));
                        
                        CHECK_CUDA_ERROR(hipMemcpy(d_outGrad, node->grad.data, bytes, hipMemcpyHostToDevice));
                        CHECK_CUDA_ERROR(hipMemcpy(d_input, node->inputs[0]->value.data, bytes, hipMemcpyHostToDevice));
                        
                        int blockSize = 256;
                        int numBlocks = (size + blockSize - 1) / blockSize;
                        
                        reluBackwardKernel<<<numBlocks, blockSize>>>(d_outGrad, d_input, d_inGrad, size);
                        
                        CHECK_CUDA_ERROR(hipGetLastError());
                        CHECK_CUDA_ERROR(hipMemcpy(node->inputs[0]->grad.data, d_inGrad, bytes, hipMemcpyDeviceToHost));
                        
                        CHECK_CUDA_ERROR(hipFree(d_outGrad));
                        CHECK_CUDA_ERROR(hipFree(d_input));
                        CHECK_CUDA_ERROR(hipFree(d_inGrad));
                    }
                    break;
                    
                case OP_SIGMOID:
                    // Sigmoid反向传播
                    {
                        const int size = node->grad.row * node->grad.col;
                        const size_t bytes = size * sizeof(float);
                        
                        float *d_outGrad, *d_output, *d_inGrad;
                        CHECK_CUDA_ERROR(hipMalloc(&d_outGrad, bytes));
                        CHECK_CUDA_ERROR(hipMalloc(&d_output, bytes));
                        CHECK_CUDA_ERROR(hipMalloc(&d_inGrad, bytes));
                        
                        CHECK_CUDA_ERROR(hipMemcpy(d_outGrad, node->grad.data, bytes, hipMemcpyHostToDevice));
                        CHECK_CUDA_ERROR(hipMemcpy(d_output, node->value.data, bytes, hipMemcpyHostToDevice));
                        
                        int blockSize = 256;
                        int numBlocks = (size + blockSize - 1) / blockSize;
                        
                        sigmoidBackwardKernel<<<numBlocks, blockSize>>>(d_outGrad, d_output, d_inGrad, size);
                        
                        CHECK_CUDA_ERROR(hipGetLastError());
                        CHECK_CUDA_ERROR(hipMemcpy(node->inputs[0]->grad.data, d_inGrad, bytes, hipMemcpyDeviceToHost));
                        
                        CHECK_CUDA_ERROR(hipFree(d_outGrad));
                        CHECK_CUDA_ERROR(hipFree(d_output));
                        CHECK_CUDA_ERROR(hipFree(d_inGrad));
                    }
                    break;
                
                // 其他操作...
                default:
                    std::cerr << "Unsupported operation in backward pass." << std::endl;
                    break;
            }
        }
    }
};
// CPU version of matrix multiplication
void matrixMatMulCPU(const Matrix_CU& A, const Matrix_CU& B, Matrix_CU& C) {
    assert(A.col == B.row);
    assert(C.row == A.row && C.col == B.col);
    
    for (int i = 0; i < A.row; ++i) {
        for (int j = 0; j < B.col; ++j) {
            C.data[i * C.col + j] = 0.0f;
            for (int k = 0; k < A.col; ++k) {
                C.data[i * C.col + j] += A.data[i * A.col + k] * B.data[k * B.col + j];
            }
        }
    }
}

int test4fullconnect() {
    // 可配置参数
    int input_rows = 100;        
    int input_cols = 100;         
    int output_cols = 50;         
    int batch_size = 32;           
    bool use_relu = true;          
    int print_rows = 5;           
    int print_cols = 5;           
    bool verify_with_cpu = true; 
    hipFree(0);
    std::cout << "====== Configuration ======\n";
    std::cout << "Input matrix: " << input_rows << " x " << input_cols << std::endl;
    std::cout << "Weight matrix: " << input_cols << " x " << output_cols << std::endl;
    std::cout << "Batch size: " << batch_size << std::endl;
    std::cout << "Activation: " << (use_relu ? "ReLU" : "None") << std::endl;
    
    // 创建计算图
    ComputeGraph graph;
    
    ComputeNode* input = graph.addInput(input_rows, input_cols, batch_size);
    ComputeNode* weight = graph.addInput(input_cols, output_cols);
    
    input->value.randomInit();
    weight->value.randomInit();
    
    // 创建CPU版本的矩阵用于验证
    Matrix_CU input_cpu = input->value;
    Matrix_CU weight_cpu = weight->value;
    
    // 记录CUDA计算开始时间
    auto cuda_start = std::chrono::high_resolution_clock::now();
    
    // 构建前向计算图
    ComputeNode* fc1 = graph.addMatMul(input, weight);  // 全连接层
    ComputeNode* output_node = fc1;
    
    // 如果需要，添加ReLU激活
    if (use_relu) {
        output_node = graph.addReLU(fc1);
    }
    
    // 计算前向传播
    graph.forward();
    
    // 反向传播计算梯度
    graph.backward(output_node);
    
    // 记录CUDA计算结束时间
    auto cuda_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> cuda_duration = cuda_end - cuda_start;
    
    // 输出CUDA计算结果
    std::cout << "\n====== CUDA Results ======\n";
    std::cout << "Input first " << print_rows << "x" << print_cols << ":\n";
    input->value.printSubMatrix(print_rows, print_cols);
    
    std::cout << "Weight first " << print_rows << "x" << print_cols << ":\n";
    weight->value.printSubMatrix(std::min(print_rows, weight->value.row), 
                                std::min(print_cols, weight->value.col));
    
    std::cout << "Output first " << print_rows << "x" << print_cols << ":\n";
    output_node->value.printSubMatrix(print_rows, 
                                     std::min(print_cols, output_node->value.col));
    
    std::cout << "Input gradient first " << print_rows << "x" << print_cols << ":\n";
    input->grad.printSubMatrix(print_rows, print_cols);
    
    std::cout << "Weight gradient first " << print_rows << "x" << print_cols << ":\n";
    weight->grad.printSubMatrix(std::min(print_rows, weight->value.row), 
                               std::min(print_cols, weight->value.col));
    
    // 如果需要，使用CPU验证
    if (verify_with_cpu) {
        std::cout << "\n====== CPU Verification ======\n";
        
        // 记录CPU计算开始时间
        auto cpu_start = std::chrono::high_resolution_clock::now();
        
        // CPU版本的矩阵乘法计算 (FC1)
        Matrix_CU fc1_cpu(input_cpu.row, weight_cpu.col);
        matrixMatMulCPU(input_cpu, weight_cpu, fc1_cpu);
        
        // CPU版本输出节点
        Matrix_CU output_cpu = fc1_cpu;
        
        // 如果需要，CPU版本的ReLU计算
        if (use_relu) {
            output_cpu = Matrix_CU(fc1_cpu.row, fc1_cpu.col);
            for (int i = 0; i < fc1_cpu.row * fc1_cpu.col; ++i) {
                output_cpu.data[i] = fc1_cpu.data[i] > 0 ? fc1_cpu.data[i] : 0;
            }
        }
        
        // CPU版本的反向传播
        // 初始化输出梯度为1.0
        Matrix_CU output_grad(output_cpu.row, output_cpu.col);
        for (int i = 0; i < output_grad.row * output_grad.col; ++i) {
            output_grad.data[i] = 1.0f;
        }
        
        // ReLU反向传播
        Matrix_CU fc1_grad = output_grad;
        if (use_relu) {
            fc1_grad = Matrix_CU(fc1_cpu.row, fc1_cpu.col);
            for (int i = 0; i < fc1_cpu.row * fc1_cpu.col; ++i) {
                fc1_grad.data[i] = fc1_cpu.data[i] > 0 ? output_grad.data[i] : 0;
            }
        }
        
        // 矩阵乘法反向传播
        Matrix_CU input_grad_cpu(input_cpu.row, input_cpu.col);
        Matrix_CU weight_grad_cpu(weight_cpu.row, weight_cpu.col);
        
        // 计算输入梯度: input_grad = fc1_grad * weight^T
        Matrix_CU weight_transpose(weight_cpu.col, weight_cpu.row);
        for (int i = 0; i < weight_cpu.row; ++i) {
            for (int j = 0; j < weight_cpu.col; ++j) {
                weight_transpose.data[j * weight_cpu.row + i] = weight_cpu.data[i * weight_cpu.col + j];
            }
        }
        matrixMatMulCPU(fc1_grad, weight_transpose, input_grad_cpu);
        
        // 计算权重梯度: weight_grad = input^T * fc1_grad
        Matrix_CU input_transpose(input_cpu.col, input_cpu.row);
        for (int i = 0; i < input_cpu.row; ++i) {
            for (int j = 0; j < input_cpu.col; ++j) {
                input_transpose.data[j * input_cpu.row + i] = input_cpu.data[i * input_cpu.col + j];
            }
        }
        matrixMatMulCPU(input_transpose, fc1_grad, weight_grad_cpu);
        
        // 记录CPU计算结束时间
        auto cpu_end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> cpu_duration = cpu_end - cpu_start;
        
        // 输出CPU计算结果
        std::cout << "CPU Output first " << print_rows << "x" << print_cols << ":\n";
        output_cpu.printSubMatrix(print_rows, std::min(print_cols, output_cpu.col));
        
        std::cout << "CPU Input gradient first " << print_rows << "x" << print_cols << ":\n";
        input_grad_cpu.printSubMatrix(print_rows, print_cols);
        
        std::cout << "CPU Weight gradient first " << print_rows << "x" << print_cols << ":\n";
        weight_grad_cpu.printSubMatrix(std::min(print_rows, weight_grad_cpu.row), 
                                      std::min(print_cols, weight_grad_cpu.col));
        
        // 验证CUDA和CPU结果的差异
        double output_diff = 0.0;
        double input_grad_diff = 0.0;
        double weight_grad_diff = 0.0;
        
        for (int i = 0; i < output_node->value.row * output_node->value.col; ++i) {
            output_diff += std::abs(output_node->value.data[i] - output_cpu.data[i]);
        }
        output_diff /= (output_node->value.row * output_node->value.col);
        
        for (int i = 0; i < input->grad.row * input->grad.col; ++i) {
            input_grad_diff += std::abs(input->grad.data[i] - input_grad_cpu.data[i]);
        }
        input_grad_diff /= (input->grad.row * input->grad.col);
        
        for (int i = 0; i < weight->grad.row * weight->grad.col; ++i) {
            weight_grad_diff += std::abs(weight->grad.data[i] - weight_grad_cpu.data[i]);
        }
        weight_grad_diff /= (weight->grad.row * weight->grad.col);
        
        // 输出结果比较和性能比较
        std::cout << "\n====== Verification Results ======\n";
        std::cout << "Average output difference: " << output_diff << std::endl;
        std::cout << "Average input gradient difference: " << input_grad_diff << std::endl;
        std::cout << "Average weight gradient difference: " << weight_grad_diff << std::endl;
        
        std::cout << "\n====== Performance Comparison ======\n";
        std::cout << "CUDA computation time: " << cuda_duration.count() << " ms" << std::endl;
        std::cout << "CPU computation time: " << cpu_duration.count() << " ms" << std::endl;
        std::cout << "Speedup: " << cpu_duration.count() / cuda_duration.count() << "x" << std::endl;
    } else {
        std::cout << "\n====== Performance ======\n";
        std::cout << "CUDA computation time: " << cuda_duration.count() << " ms" << std::endl;
    }
    
    return 0;
}