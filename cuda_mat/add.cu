#include <iostream>
#include <cstdlib>
#include <chrono>
#include <hip/hip_runtime.h>

// CUDA 核函数：向量加法
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

int main()
{
    // 设置向量大小 (1百万个元素)
    int numElements = 1 << 20;
    size_t size = numElements * sizeof(float);
    std::cout << "向量加法测试，每个向量包含 " << numElements << " 个元素\n";

    // 分配主机内存
    float *h_A = new float[numElements];
    float *h_B = new float[numElements];
    float *h_C = new float[numElements];
    float *d_A = nullptr;
    float *d_B = nullptr;
    float *d_C = nullptr;

    // 初始化输入向量
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // 分配设备内存
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // 拷贝数据到设备
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // 启动 CUDA 核函数
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    std::cout << "CUDA 核函数配置: " << blocksPerGrid << " 个块, 每个块 " << threadsPerBlock << " 个线程\n";

    // 测试 GPU 性能
    auto start_gpu = std::chrono::high_resolution_clock::now();
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    hipDeviceSynchronize(); // 等待 GPU 完成
    auto end_gpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> gpu_time = end_gpu - start_gpu;
    std::cout << "GPU 执行时间: " << gpu_time.count() * 1000 << " 毫秒\n";

    // 拷贝结果回主机
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // 测试 CPU 性能
    auto start_cpu = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < numElements; ++i)
    {
        h_C[i] = h_A[i] + h_B[i];
    }
    auto end_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> cpu_time = end_cpu - start_cpu;
    std::cout << "CPU 执行时间: " << cpu_time.count() * 1000 << " 毫秒\n";

    // 验证结果
    for (int i = 0; i < numElements; i++)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            std::cerr << "结果验证失败!\n";
            break;
        }
    }

    // 释放内存
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    std::cout << "测试完成\n";

    return 0;
}